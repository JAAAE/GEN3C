#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/** @file   testbed.cu
 *  @author Thomas Müller & Alex Evans, NVIDIA
 */

#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/json_binding.h>
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/testbed.h>

#include <tiny-cuda-nn/common_host.h>

#include <json/json.hpp>

#include <filesystem/directory.h>
#include <filesystem/path.h>

#include <playne-equivalence/playne_equivalence.cuh>

#include <fstream>
#include <unordered_set>

#ifdef NGP_GUI
#	include <imgui/backends/imgui_impl_glfw.h>
#	include <imgui/backends/imgui_impl_opengl3.h>
#	include <imgui/misc/cpp/imgui_stdlib.h>
#	include <imgui/imgui.h>
#	include <imguizmo/ImGuizmo.h>
#	ifdef _WIN32
#		include <GL/gl3w.h>
#	else
#		include <GL/glew.h>
#	endif
#	include <GLFW/glfw3.h>
#	include <GLFW/glfw3native.h>
#	include <cuda_gl_interop.h>
#endif

// Windows.h is evil
#undef min
#undef max
#undef near
#undef far

using namespace std::literals::chrono_literals;
using nlohmann::json;

namespace ngp {

int do_system(const std::string& cmd) {
#ifdef _WIN32
	tlog::info() << "> " << cmd;
	return _wsystem(utf8_to_utf16(cmd).c_str());
#else
	tlog::info() << "$ " << cmd;
	return system(cmd.c_str());
#endif
}

std::atomic<size_t> g_total_n_bytes_allocated{0};

void Testbed::update_imgui_paths() {
	snprintf(m_imgui.cam_path_path, sizeof(m_imgui.cam_path_path), "%s", (root_dir() / "cam.json").str().c_str());
	snprintf(m_imgui.video_path, sizeof(m_imgui.video_path), "%s", (root_dir() / "video.json").str().c_str());
	snprintf(m_imgui.cam_export_path, sizeof(m_imgui.cam_export_path), "%s", (root_dir() / "cam_export.json").str().c_str());
}

void Testbed::set_mode(ETestbedMode mode) {
	if (mode == m_testbed_mode) {
		return;
	}

	// Clear device-owned data that might be mode-specific
	for (auto&& device : m_devices) {
		device.clear();
	}

	m_testbed_mode = mode;

	// Set various defaults depending on mode
	m_use_aux_devices = false;

	if (m_testbed_mode == ETestbedMode::Gen3c) {
		if (m_dlss_provider && m_aperture_size == 0.0f) {
			m_dlss = true;
		}
	} else {
		m_dlss = false;
	}

	m_reproject_enable = m_testbed_mode == ETestbedMode::Gen3c;

	reset_camera();

#ifdef NGP_GUI
	update_vr_performance_settings();
#endif
}

void Testbed::load_file(const fs::path& path) {
	if (!path.exists()) {
		tlog::error() << "File '" << path.str() << "' does not exist.";
		return;
	}

	// If we get a json file, we need to parse it to determine its purpose.
	if (equals_case_insensitive(path.extension(), "json")) {
		json file;
		{
			std::ifstream f{native_string(path)};
			file = json::parse(f, nullptr, true, true);
		}

		// Camera path
		if (file.contains("path")) {
			load_camera_path(path);
			return;
		}
	}

	tlog::error() << "File '" << path.str() << "' is not a valid file to load.";
}

void Testbed::reset_accumulation(bool due_to_camera_movement, bool immediate_redraw, bool reset_pip) {
	if (immediate_redraw) {
		redraw_next_frame();
	}

	if (!due_to_camera_movement || !reprojection_available()) {
		m_windowless_render_surface.reset_accumulation();
		for (auto& view : m_views) {
			view.render_buffer->reset_accumulation();
		}
	}

	if (reset_pip) {
		m_pip_render_buffer->reset_accumulation();
	}
}

void Testbed::translate_camera(const vec3& rel, const mat3& rot, bool allow_up_down) {
	vec3 movement = rot * rel;
	if (!allow_up_down) {
		movement -= dot(movement, m_up_dir) * m_up_dir;
	}

	m_camera[3] += movement;
	reset_accumulation(true);
}

vec3 Testbed::look_at() const { return view_pos() + view_dir() * m_scale; }

void Testbed::set_look_at(const vec3& pos) { m_camera[3] += pos - look_at(); }

void Testbed::set_scale(float scale) {
	auto prev_look_at = look_at();
	m_camera[3] = (view_pos() - prev_look_at) * (scale / m_scale) + prev_look_at;
	m_scale = scale;
}

void Testbed::set_view_dir(const vec3& dir) {
	auto old_look_at = look_at();
	m_camera[0] = normalize(cross(dir, m_up_dir));
	m_camera[1] = normalize(cross(dir, m_camera[0]));
	m_camera[2] = normalize(dir);
	set_look_at(old_look_at);
}

void Testbed::reset_camera() {
	m_fov_axis = 1;
	m_zoom = 1.0f;
	m_screen_center = vec2(0.5f);

	set_fov(50.625f);
	m_scale = 1.5f;

	m_camera = m_default_camera;
	m_camera[3] -= m_scale * view_dir();

	m_smoothed_camera = m_camera;
	m_sun_dir = normalize(vec3(1.0f));

	reset_accumulation();
}

fs::path Testbed::root_dir() {
	if (m_root_dir.empty()) {
		set_root_dir(discover_root_dir());
	}

	return m_root_dir;
}

void Testbed::set_root_dir(const fs::path& dir) { m_root_dir = dir; }

inline float linear_to_db(float x) { return -10.f * logf(x) / logf(10.f); }


#ifdef NGP_GUI
bool imgui_colored_button(const char* name, float hue) {
	ImGui::PushStyleColor(ImGuiCol_Button, (ImVec4)ImColor::HSV(hue, 0.6f, 0.6f));
	ImGui::PushStyleColor(ImGuiCol_ButtonHovered, (ImVec4)ImColor::HSV(hue, 0.7f, 0.7f));
	ImGui::PushStyleColor(ImGuiCol_ButtonActive, (ImVec4)ImColor::HSV(hue, 0.8f, 0.8f));
	bool rv = ImGui::Button(name);
	ImGui::PopStyleColor(3);
	return rv;
}

void Testbed::overlay_fps() {
	ImGui::PushFont((ImFont*)m_imgui.overlay_font);
	ImGui::SetNextWindowPos({10.0f, 10.0f}, ImGuiCond_Always, {0.0f, 0.0f});
	ImGui::SetNextWindowBgAlpha(0.35f);
	if (ImGui::Begin(
			"Overlay",
			nullptr,
			ImGuiWindowFlags_NoDecoration | ImGuiWindowFlags_AlwaysAutoResize | ImGuiWindowFlags_NoSavedSettings |
				ImGuiWindowFlags_NoFocusOnAppearing | ImGuiWindowFlags_NoNav | ImGuiWindowFlags_NoMove
		)) {
		ImGui::Text("%.1f FPS", 1000.0f / m_render_ms.ema_val());
	}
	ImGui::PopFont();
}

void Testbed::imgui() {
	// If a GUI interaction causes an error, write that error to the following string and call
	//   ImGui::OpenPopup("Error");
	static std::string imgui_error_string = "";

	m_picture_in_picture_res = 0;

	// Good default position and size for the camera path editing window
	ImGui::SetNextWindowPos({10.0f, 10.0f}, ImGuiCond_FirstUseEver);
	int window_width, window_height;
	glfwGetWindowSize(m_glfw_window, &window_width, &window_height);
	ImGui::SetNextWindowSize({420.0f, window_height - 20.0f}, ImGuiCond_FirstUseEver);

	if (ImGui::Begin("Camera path & video generation", 0, ImGuiWindowFlags_NoScrollbar)) {
		if (ImGui::CollapsingHeader("Path manipulation", ImGuiTreeNodeFlags_DefaultOpen)) {
			ImGui::Checkbox("Record camera path", &m_record_camera_path);
			ImGui::SameLine();
			if (ImGui::Button("Clear")) {
				m_camera_path.clear();
			}

			if (m_reproject_enable) {
				ImGui::SameLine();
				if (ImGui::Button("Init from views")) {
					init_camera_path_from_reproject_src_cameras();
				}
			}

			if (int read = m_camera_path.imgui(m_imgui.cam_path_path, m_frame_ms.val(), m_camera, fov(), mat4x3::identity())) {
				if (!m_camera_path.rendering || m_gen3c_render_with_gen3c) {
					reset_accumulation(true);

					if (m_camera_path.update_cam_from_path) {
						set_camera_from_time(m_camera_path.play_time);

						// A value of larger than 1 indicates that the camera path wants
						// to override camera smoothing.
						if (read > 1) {
							m_smoothed_camera = m_camera;
						}
					} else {
						m_pip_render_buffer->reset_accumulation();
					}
				}
			}

			if (!m_camera_path.keyframes.empty()) {
				float w = ImGui::GetContentRegionAvail().x;
				if (m_camera_path.update_cam_from_path) {
					m_picture_in_picture_res = 0;
					ImGui::Image((ImTextureID)(size_t)m_rgba_render_textures.front()->texture(), ImVec2(w, w * 9.0f / 16.0f));
				} else {
					m_picture_in_picture_res = (float)std::min((int(w) + 31) & (~31), 1920 / 4);
					ImGui::Image((ImTextureID)(size_t)m_pip_render_texture->texture(), ImVec2(w, w * 9.0f / 16.0f));
				}
			}
		}

		if (!m_camera_path.keyframes.empty() && ImGui::CollapsingHeader("Video generation", ImGuiTreeNodeFlags_DefaultOpen)) {
			// Render a video
			// TODO: simplify this (only allow rendering with Gen3C).
			ImGui::BeginDisabled(m_camera_path.rendering);
			if (imgui_colored_button(m_camera_path.rendering ? "Waiting for model..." : "Generate video", 0.4)) {
				bool was_rendering = m_camera_path.rendering;
				m_camera_path.rendering = !m_camera_path.rendering;

				if (m_gen3c_render_with_gen3c) {
					if (m_gen3c_cb) {
						m_gen3c_cb(was_rendering ? "abort_inference" : "request_inference");
					}
				} else {
					if (!clear_tmp_dir()) {
						imgui_error_string = "Failed to clear temporary directory 'tmp' to hold rendered images.";
						ImGui::OpenPopup("Error");

						m_camera_path.rendering = false;
					}

					if (m_camera_path.rendering) {
						m_camera_path.render_start_time = std::chrono::steady_clock::now();
						m_camera_path.update_cam_from_path = true;
						m_camera_path.play_time = 0.0f;
						m_camera_path.auto_play_speed = 1.0f;
						m_camera_path.render_frame_idx = 0;

						m_dlss = false;

						reset_accumulation(true);
						set_camera_from_time(m_camera_path.play_time);
						m_smoothed_camera = m_camera;
					} else {
						m_camera_path.update_cam_from_path = false;
						m_camera_path.play_time = 0.0f;
						m_camera_path.auto_play_speed = 0.0f;
					}
				}
			}
			ImGui::EndDisabled();

			ImGui::SameLine();
			ImGui::BeginDisabled(!m_gen3c_inference_is_connected || !m_gen3c_cb);
			ImGui::Checkbox("Gen3C inference", &m_gen3c_render_with_gen3c);
			ImGui::EndDisabled();

			if (m_camera_path.rendering) {
				const auto elapsed = std::chrono::steady_clock::now() - m_camera_path.render_start_time;

				const float duration = m_camera_path.duration_seconds();
				const uint32_t progress = m_camera_path.render_frame_idx * m_camera_path.render_settings.spp + m_views.front().render_buffer->spp();
				const uint32_t goal = m_camera_path.render_settings.n_frames(duration) * m_camera_path.render_settings.spp;
				const auto est_remaining = elapsed * (float)(goal - progress) / std::max(progress, 1u);

				if (m_gen3c_render_with_gen3c) {
					if (!m_gen3c_inference_info.empty()) {
						ImGui::TextWrapped("%s", m_gen3c_inference_info.c_str());
					}

					if (m_gen3c_inference_progress > 0) {
						ImGui::ProgressBar(m_gen3c_inference_progress);
					}
				} else {
					ImGui::Text(
						"%s",
						fmt::format(
							"Frame {}/{}, Elapsed: {}, Remaining: {}",
							m_camera_path.render_frame_idx + 1,
							m_camera_path.render_settings.n_frames(duration),
							tlog::durationToString(std::chrono::steady_clock::now() - m_camera_path.render_start_time),
							tlog::durationToString(est_remaining)
						)
							.c_str()
					);

					ImGui::ProgressBar((float)progress / goal);
				}
			}

			ImGui::BeginDisabled(m_camera_path.rendering);

			ImGui::Checkbox("Show rendered Gen3C cache in video", &m_gen3c_show_cache_renderings);
			// Note: 3D cache visualization is incompatible with adding Gen3C frames to the viewport.
			if (m_gen3c_show_cache_renderings)
				m_gen3c_display_frames = false;
			ImGui::BeginDisabled(m_gen3c_show_cache_renderings);
			ImGui::Checkbox("Add Gen3C keyframes to viewport after inference", &m_gen3c_display_frames);
			ImGui::EndDisabled();  // m_gen3c_show_cache_renderings

			ImGui::InputText("Video file##Video file path", m_imgui.video_path, sizeof(m_imgui.video_path));
			m_camera_path.render_settings.filename = m_imgui.video_path;
			ImGui::SliderInt("MP4 quality", &m_camera_path.render_settings.quality, 0, 10);

			float duration_seconds = m_camera_path.duration_seconds();
			if (ImGui::InputFloat("Duration (seconds)", &duration_seconds) && duration_seconds > 0.0f) {
				m_camera_path.set_duration_seconds(duration_seconds);
			}

			ImGui::InputFloat("FPS (frames/second)", &m_camera_path.render_settings.fps);

			ImGui::BeginDisabled(m_gen3c_render_with_gen3c);
			ImGui::InputInt2("Resolution", &m_camera_path.render_settings.resolution.x);
			// ImGui::InputInt("SPP (samples/pixel)", &m_camera_path.render_settings.spp);
			if (m_gen3c_render_with_gen3c) {
				m_camera_path.render_settings.spp = 1;
			}
			// ImGui::SliderFloat("Shutter fraction", &m_camera_path.render_settings.shutter_fraction, 0.0f, 1.0f);
			ImGui::EndDisabled(); // end m_gen3c_render_with_gen3c

			ImGui::EndDisabled(); // end m_camera_path.rendering

			ImGui::Spacing();
			bool export_cameras = imgui_colored_button("Export cameras", 0.7);

			ImGui::SameLine();

			static bool w2c = false;
			ImGui::Checkbox("W2C", &w2c);

			ImGui::InputText("Cameras file##Camera export path", m_imgui.cam_export_path, sizeof(m_imgui.cam_export_path));
			m_camera_path.render_settings.filename = m_imgui.video_path;

			if (export_cameras) {
				std::vector<json> cameras;
				const float duration = m_camera_path.duration_seconds();
				for (uint32_t i = 0; i < m_camera_path.render_settings.n_frames(duration); ++i) {
					mat4x3 start_cam = m_camera_path.eval_camera_path((float)i / (m_camera_path.render_settings.n_frames(duration))).m();
					mat4x3 end_cam = m_camera_path
										 .eval_camera_path(
											 ((float)i + m_camera_path.render_settings.shutter_fraction) /
											 (m_camera_path.render_settings.n_frames(duration))
										 )
										 .m();
					if (w2c) {
						start_cam = inverse(mat4x4(start_cam));
						end_cam = inverse(mat4x4(end_cam));
					}

					cameras.push_back({
						{"start", start_cam},
						{"end",   end_cam  },
					});
				}

				json j;
				j["cameras"] = cameras;
				j["resolution"] = m_camera_path.render_settings.resolution;
				j["duration_seconds"] = m_camera_path.duration_seconds();
				j["fps"] = m_camera_path.render_settings.fps;
				j["spp"] = m_camera_path.render_settings.spp;
				j["quality"] = m_camera_path.render_settings.quality;
				j["shutter_fraction"] = m_camera_path.render_settings.shutter_fraction;

				std::ofstream f(native_string(m_imgui.cam_export_path));
				f << j;
			}
		}
	}
	ImGui::End();

	// Good default position and size for the right-hand side window
	int pane_width = 350;
	ImGui::SetNextWindowPos({window_width - pane_width - 10.0f, 10.0f}, ImGuiCond_FirstUseEver);
	ImGui::SetNextWindowSize({(float)pane_width, window_height - 20.0f}, ImGuiCond_FirstUseEver);

	ImGui::Begin("Gen3C v" NGP_VERSION);

	size_t n_bytes = tcnn::total_n_bytes_allocated() + g_total_n_bytes_allocated;
	if (m_dlss_provider) {
		n_bytes += m_dlss_provider->allocated_bytes();
	}

	ImGui::Text("Frame: %.2f ms (%.1f FPS); Mem: %s", m_frame_ms.ema_val(), 1000.0f / m_frame_ms.ema_val(), bytes_to_string(n_bytes).c_str());
	bool accum_reset = false;

	if (m_testbed_mode == ETestbedMode::Gen3c && ImGui::CollapsingHeader("Video generation server", ImGuiTreeNodeFlags_DefaultOpen)) {
		ImGui::TextWrapped("%s", m_gen3c_info.c_str());
		ImGui::Spacing();

		// Create a child box with a title and borders
		if (ImGui::TreeNodeEx("Seeding", ImGuiTreeNodeFlags_DefaultOpen)) {
			ImGui::TextWrapped("Enter the path to an image or a pre-processed video directory.");
			ImGui::InputText("Path", &m_gen3c_seed_path);

			ImGui::BeginDisabled(m_gen3c_seed_path.empty());
			if (ImGui::Button("Seed") && m_gen3c_cb) {
				m_gen3c_cb("seed_model");
			}
			if (m_gen3c_seeding_progress > 0) {
				ImGui::ProgressBar(m_gen3c_seeding_progress);
			}
			ImGui::EndDisabled();

			ImGui::Spacing();
			ImGui::TreePop();
		}

		// ImGui::Separator();

		// We need this to be executed even if the panel below is collapsed.
		switch (m_gen3c_camera_source) {
			case EGen3cCameraSource::Fake: {
				m_gen3c_auto_inference = false;
				break;
			}
			case EGen3cCameraSource::Viewpoint: {
				break;
			}
			case EGen3cCameraSource::Authored: {
				m_gen3c_auto_inference = false;
				break;
			}
			default: throw std::runtime_error("Unsupported Gen3C camera source.");
		}

	}

	if (ImGui::CollapsingHeader("Point cloud", ImGuiTreeNodeFlags_DefaultOpen)) {
		// accum_reset |= ImGui::Checkbox("Enable reprojection", &m_reproject_enable);
		if (m_reproject_enable) {
			int max_views = (int)m_reproject_src_views.size();

			int prev_min_src_view_index = m_reproject_min_src_view_index;
			int prev_max_src_view_index = m_reproject_max_src_view_index;
			int prev_n_frames_shown = std::max(0, prev_max_src_view_index - prev_min_src_view_index);

			if (ImGui::SliderInt("Min view index", &m_reproject_min_src_view_index, 0, max_views)) {
				// If shift, move the range synchronously.
				if (ImGui::GetIO().KeyShift) {
					m_reproject_max_src_view_index =
						std::min(m_reproject_max_src_view_index + m_reproject_min_src_view_index - prev_min_src_view_index, max_views);
					// Keep the number of frames shown constant.
					m_reproject_min_src_view_index = m_reproject_max_src_view_index - prev_n_frames_shown;
				}

				// Ensure that range remains valid (max index >= min index).
				m_reproject_max_src_view_index = std::max(m_reproject_max_src_view_index, m_reproject_min_src_view_index);
				accum_reset = true;
			}

			if (ImGui::SliderInt("Max view index", &m_reproject_max_src_view_index, 0, max_views)) {
				// If shift, move the range synchronously.
				if (ImGui::GetIO().KeyShift) {
					m_reproject_min_src_view_index =
						std::max(m_reproject_min_src_view_index + m_reproject_max_src_view_index - prev_max_src_view_index, 0);
					// Keep the number of frames shown constant.
					m_reproject_max_src_view_index = m_reproject_min_src_view_index + prev_n_frames_shown;
				}
				// Ensure that range remains valid (max index >= min index).
				m_reproject_min_src_view_index = std::min(m_reproject_max_src_view_index, m_reproject_min_src_view_index);
				accum_reset = true;
			}

			if (max_views > 0 && ImGui::SliderInt("Snap to view", (int*)&m_reproject_selected_src_view, 0, max_views - 1)) {
				m_camera = m_smoothed_camera =
					m_reproject_src_views[std::min((size_t)m_reproject_selected_src_view, m_reproject_src_views.size() - 1)].camera0;
				accum_reset = true;
			}

			accum_reset |= ImGui::Checkbox("Visualize views", &m_reproject_visualize_src_views);
			ImGui::SameLine();
			if (ImGui::Button("Delete views")) {
				clear_src_views();
			}

			if (ImGui::TreeNodeEx("Advanced reprojection settings")) {
				accum_reset |= ImGui::SliderFloat(
					"Reproject min t", &m_reproject_min_t, 0.01f, 16.0f, "%.01f", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat
				);
				accum_reset |= ImGui::SliderFloat(
					"Reproject scaling", &m_reproject_step_factor, 1.003f, 1.5f, "%.001f", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat
				);

				accum_reset |= ImGui::Combo("Reproject render mode", (int*)&m_pm_viz_mode, PmVizModeStr);

				ImGui::TreePop();
			}

		}
	}

	if (ImGui::CollapsingHeader("Rendering", m_testbed_mode == ETestbedMode::Gen3c ? 0 : ImGuiTreeNodeFlags_DefaultOpen)) {

		ImGui::Checkbox("Render", &m_render);
		ImGui::SameLine();

		const auto& render_buffer = m_views.front().render_buffer;
		std::string spp_string = m_dlss ? std::string{""} : fmt::format("({} spp)", std::max(render_buffer->spp(), 1u));
		ImGui::Text(
			": %.01fms for %dx%d %s",
			m_render_ms.ema_val(),
			render_buffer->in_resolution().x,
			render_buffer->in_resolution().y,
			spp_string.c_str()
		);

		ImGui::SameLine();
		if (ImGui::Checkbox("VSync", &m_vsync)) {
			glfwSwapInterval(m_vsync ? 1 : 0);
		}


		ImGui::Checkbox("Dynamic resolution", &m_dynamic_res);
		ImGui::SameLine();
		ImGui::PushItemWidth(ImGui::GetWindowWidth() * 0.3f);
		if (m_dynamic_res) {
			ImGui::SliderFloat(
				"Target FPS", &m_dynamic_res_target_fps, 2.0f, 144.0f, "%.01f", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat
			);
		} else {
			ImGui::SliderInt("Resolution factor", &m_fixed_res_factor, 8, 64);
		}
		ImGui::PopItemWidth();

		if (ImGui::TreeNode("Advanced rendering options")) {
			accum_reset |= ImGui::Combo("Render mode", (int*)&m_render_mode, RenderModeStr);
			accum_reset |= ImGui::Combo("Tonemap curve", (int*)&m_tonemap_curve, TonemapCurveStr);
			accum_reset |= ImGui::ColorEdit4("Background", &m_background_color[0]);

			if (ImGui::SliderFloat("Exposure", &m_exposure, -5.f, 5.f)) {
				set_exposure(m_exposure);
			}

			ImGui::SliderInt("Max spp", &m_max_spp, 0, 1024, "%d", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat);
			accum_reset |= ImGui::Checkbox("Render transparency as checkerboard", &m_render_transparency_as_checkerboard);
			accum_reset |= ImGui::Combo("Color space", (int*)&m_color_space, ColorSpaceStr);
			accum_reset |= ImGui::Checkbox("Snap to pixel centers", &m_snap_to_pixel_centers);

			ImGui::TreePop();
		}
	}

	if (ImGui::CollapsingHeader("Camera")) {
		ImGui::Checkbox("First person controls", &m_fps_camera);
		ImGui::SameLine();
		ImGui::Checkbox("Smooth motion", &m_camera_smoothing);

		float local_fov = fov();
		if (ImGui::SliderFloat("Field of view", &local_fov, 0.0f, 120.0f)) {
			set_fov(local_fov);
			accum_reset = true;
		}

		if (ImGui::TreeNode("Advanced camera settings")) {
			accum_reset |= ImGui::SliderFloat2("Screen center", &m_screen_center.x, 0.f, 1.f);
			accum_reset |= ImGui::SliderFloat2("Parallax shift", &m_parallax_shift.x, -1.f, 1.f);
			accum_reset |= ImGui::SliderFloat("Slice / focus depth", &m_slice_plane_z, -m_bounding_radius, m_bounding_radius);
			accum_reset |= ImGui::SliderFloat(
				"Render near distance", &m_render_near_distance, 0.0f, 1.0f, "%.3f", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat
			);

			bool lens_changed = ImGui::Checkbox("Apply lens distortion", &m_render_with_lens_distortion);
			if (m_render_with_lens_distortion) {
				lens_changed |= ImGui::Combo("Lens mode", (int*)&m_render_lens.mode, LensModeStr);
				if (m_render_lens.mode == ELensMode::OpenCV) {
					accum_reset |= ImGui::InputFloat("k1", &m_render_lens.params[0], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("k2", &m_render_lens.params[1], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("p1", &m_render_lens.params[2], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("p2", &m_render_lens.params[3], 0.f, 0.f, "%.5f");
				} else if (m_render_lens.mode == ELensMode::OpenCVFisheye) {
					accum_reset |= ImGui::InputFloat("k1", &m_render_lens.params[0], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("k2", &m_render_lens.params[1], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("k3", &m_render_lens.params[2], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("k4", &m_render_lens.params[3], 0.f, 0.f, "%.5f");
				} else if (m_render_lens.mode == ELensMode::FTheta) {
					accum_reset |= ImGui::InputFloat("width", &m_render_lens.params[5], 0.f, 0.f, "%.0f");
					accum_reset |= ImGui::InputFloat("height", &m_render_lens.params[6], 0.f, 0.f, "%.0f");
					accum_reset |= ImGui::InputFloat("f_theta p0", &m_render_lens.params[0], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("f_theta p1", &m_render_lens.params[1], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("f_theta p2", &m_render_lens.params[2], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("f_theta p3", &m_render_lens.params[3], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("f_theta p4", &m_render_lens.params[4], 0.f, 0.f, "%.5f");
				}

				if (lens_changed && !m_render_lens.supports_dlss()) {
					m_dlss = false;
				}
			}
			ImGui::Spacing();

			accum_reset |= lens_changed;

			char buf[2048];
			vec3 v = view_dir();
			vec3 p = look_at();
			vec3 s = m_sun_dir;
			vec3 u = m_up_dir;
			vec4 b = m_background_color;
			snprintf(
				buf,
				sizeof(buf),
				"testbed.background_color = [%0.3f, %0.3f, %0.3f, %0.3f]\n"
				"testbed.exposure = %0.3f\n"
				"testbed.sun_dir = [%0.3f,%0.3f,%0.3f]\n"
				"testbed.up_dir = [%0.3f,%0.3f,%0.3f]\n"
				"testbed.view_dir = [%0.3f,%0.3f,%0.3f]\n"
				"testbed.look_at = [%0.3f,%0.3f,%0.3f]\n"
				"testbed.scale = %0.3f\n"
				"testbed.fov,testbed.aperture_size,testbed.slice_plane_z = %0.3f,%0.3f,%0.3f\n"
				"testbed.autofocus_target = [%0.3f,%0.3f,%0.3f]\n"
				"testbed.autofocus = %s\n\n",
				b.r,
				b.g,
				b.b,
				b.a,
				m_exposure,
				s.x,
				s.y,
				s.z,
				u.x,
				u.y,
				u.z,
				v.x,
				v.y,
				v.z,
				p.x,
				p.y,
				p.z,
				scale(),
				fov(),
				m_aperture_size,
				m_slice_plane_z,
				m_autofocus_target.x,
				m_autofocus_target.y,
				m_autofocus_target.z,
				m_autofocus ? "True" : "False"
			);

			ImGui::InputTextMultiline("Params", buf, sizeof(buf));
			ImGui::TreePop();
		}
	}

	if (ImGui::BeginPopupModal("Error", NULL, ImGuiWindowFlags_AlwaysAutoResize)) {
		ImGui::Text("%s", imgui_error_string.c_str());
		if (ImGui::Button("OK", ImVec2(120, 0))) {
			ImGui::CloseCurrentPopup();
		}
		ImGui::EndPopup();
	}

	if (accum_reset) {
		reset_accumulation();
	}

	if (ImGui::Button("Go to Python REPL")) {
		m_want_repl = true;
	}

	ImGui::End();
}

void Testbed::init_camera_path_from_reproject_src_cameras() {
	m_camera_path.clear();

	for (int i = m_reproject_min_src_view_index; i < std::min(m_reproject_max_src_view_index, (int)m_reproject_src_views.size()); ++i) {
		const auto& view = m_reproject_src_views[i];
		m_camera_path.add_camera(
			view.camera0,
			view.fov()[m_fov_axis],
			0.0f // timestamp set to zero: camera path treats keyframes as temporally equidistant
		);
	}

	m_camera_path.keyframe_subsampling = (int)m_camera_path.keyframes.size();
	m_camera_path.editing_kernel_type = EEditingKernel::Gaussian;
}

void Testbed::visualize_reproject_src_cameras(ImDrawList* list, const mat4& world2proj) {
	for (size_t i = (size_t)m_reproject_min_src_view_index;
		 i < std::min((size_t)m_reproject_max_src_view_index, m_reproject_src_views.size());
		 ++i) {
		const auto& view = m_reproject_src_views[i];
		auto res = view.full_resolution;
		float aspect = float(res.x) / float(res.y);

		visualize_camera(list, world2proj, view.camera0, aspect, 0xffffffff);
	}
}

void Testbed::clear_src_views() {
	m_reproject_src_views.clear();
	reset_accumulation();
}

void Testbed::draw_visualizations(ImDrawList* list, const mat4x3& camera_matrix) {
	mat4 view2world = camera_matrix;
	mat4 world2view = inverse(view2world);

	auto focal = calc_focal_length(ivec2(1), m_relative_focal_length, m_fov_axis, m_zoom);
	float zscale = 1.0f / focal[m_fov_axis];

	float xyscale = (float)m_window_res[m_fov_axis];
	vec2 screen_center = render_screen_center(m_screen_center);
	mat4 view2proj = transpose(
		mat4{
			xyscale,
			0.0f,
			(float)m_window_res.x * screen_center.x * zscale,
			0.0f,
			0.0f,
			xyscale,
			(float)m_window_res.y * screen_center.y * zscale,
			0.0f,
			0.0f,
			0.0f,
			1.0f,
			0.0f,
			0.0f,
			0.0f,
			zscale,
			0.0f,
		}
	);

	mat4 world2proj = view2proj * world2view;
	float aspect = (float)m_window_res.x / (float)m_window_res.y;

	if (m_reproject_visualize_src_views) {
		visualize_reproject_src_cameras(list, world2proj);
	}

	if (m_visualize_unit_cube) {
		visualize_cube(list, world2proj, vec3(0.f), vec3(1.f), mat3::identity());
	}

	if (m_edit_render_aabb) {
		ImGuiIO& io = ImGui::GetIO();
		// float flx = focal.x;
		float fly = focal.y;
		float zfar = m_ndc_zfar;
		float znear = m_ndc_znear;
		mat4 view2proj_guizmo = transpose(
			mat4{
				fly * 2.0f / aspect,
				0.0f,
				0.0f,
				0.0f,
				0.0f,
				-fly * 2.f,
				0.0f,
				0.0f,
				0.0f,
				0.0f,
				(zfar + znear) / (zfar - znear),
				-(2.0f * zfar * znear) / (zfar - znear),
				0.0f,
				0.0f,
				1.0f,
				0.0f,
			}
		);

		ImGuizmo::SetRect(0, 0, io.DisplaySize.x, io.DisplaySize.y);

		static mat4 matrix = mat4::identity();
		static mat4 world2view_guizmo = mat4::identity();

		vec3 cen = transpose(m_render_aabb_to_local) * m_render_aabb.center();
		if (!ImGuizmo::IsUsing()) {
			// The the guizmo is being used, it handles updating its matrix on its own.
			// Outside interference can only lead to trouble.
			auto rot = transpose(m_render_aabb_to_local);
			matrix = mat4(mat4x3(rot[0], rot[1], rot[2], cen));

			// Additionally, the world2view transform must stay fixed, else the guizmo will incorrectly
			// interpret the state from past frames. Special handling is necessary here, because below
			// we emulate world translation and rotation through (inverse) camera movement.
			world2view_guizmo = world2view;
		}

		auto prev_matrix = matrix;

		if (ImGuizmo::Manipulate(
				(const float*)&world2view_guizmo, (const float*)&view2proj_guizmo, m_camera_path.m_gizmo_op, ImGuizmo::LOCAL, (float*)&matrix, NULL, NULL
			)) {
			if (m_edit_world_transform) {
				// We transform the world by transforming the camera in the opposite direction.
				auto rel = prev_matrix * inverse(matrix);
				m_camera = mat3(rel) * m_camera;
				m_camera[3] += rel[3].xyz();

				m_up_dir = mat3(rel) * m_up_dir;
			} else {
				m_render_aabb_to_local = transpose(mat3(matrix));
				vec3 new_cen = m_render_aabb_to_local * matrix[3].xyz();
				vec3 old_cen = m_render_aabb.center();
				m_render_aabb.min += new_cen - old_cen;
				m_render_aabb.max += new_cen - old_cen;
			}

			reset_accumulation();
		}
	}


	if (m_camera_path.imgui_viz(
			list,
			view2proj,
			world2proj,
			world2view,
			focal,
			aspect,
			m_ndc_znear,
			m_ndc_zfar
		)) {
		m_pip_render_buffer->reset_accumulation();
	}
}

void glfw_error_callback(int error, const char* description) { tlog::error() << "GLFW error #" << error << ": " << description; }

bool Testbed::keyboard_event() {
	if (ImGui::GetIO().WantCaptureKeyboard) {
		return false;
	}

	if (m_keyboard_event_callback && m_keyboard_event_callback()) {
		return false;
	}

	if (ImGui::IsKeyPressed('Q') && ImGui::GetIO().KeyCtrl) {
		glfwSetWindowShouldClose(m_glfw_window, GLFW_TRUE);
	}

	if ((ImGui::IsKeyPressed(GLFW_KEY_TAB) || ImGui::IsKeyPressed(GLFW_KEY_GRAVE_ACCENT)) && !ImGui::GetIO().KeyCtrl) {
		m_imgui.mode = (ImGuiMode)(((uint32_t)m_imgui.mode + 1) % (uint32_t)ImGuiMode::NumModes);
	}

	for (int idx = 0; idx < std::min((int)ERenderMode::NumRenderModes, 10); ++idx) {
		char c[] = {"1234567890"};
		if (ImGui::IsKeyPressed(c[idx])) {
			m_render_mode = (ERenderMode)idx;
			reset_accumulation();
		}
	}

	bool ctrl = ImGui::GetIO().KeyCtrl;
	bool shift = ImGui::GetIO().KeyShift;

	if (ImGui::IsKeyPressed('Z')) {
		m_camera_path.m_gizmo_op = ImGuizmo::TRANSLATE;
	}

	if (ImGui::IsKeyPressed('X')) {
		m_camera_path.m_gizmo_op = ImGuizmo::ROTATE;
	}

	if (ImGui::IsKeyPressed('E')) {
		set_exposure(m_exposure + (shift ? -0.5f : 0.5f));
		redraw_next_frame();
	}

	if (ImGui::IsKeyPressed('R')) {
		reset_camera();
	}

	if (ImGui::IsKeyPressed('=') || ImGui::IsKeyPressed('+')) {
		if (m_fps_camera) {
			m_camera_velocity *= 1.5f;
		} else {
			set_scale(m_scale * 1.1f);
		}
	}

	if (ImGui::IsKeyPressed('-') || ImGui::IsKeyPressed('_')) {
		if (m_fps_camera) {
			m_camera_velocity /= 1.5f;
		} else {
			set_scale(m_scale / 1.1f);
		}
	}

	// WASD camera movement
	vec3 translate_vec = vec3(0.0f);
	if (ImGui::IsKeyDown('W')) {
		translate_vec.z += 1.0f;
	}

	if (ImGui::IsKeyDown('A')) {
		translate_vec.x += -1.0f;
	}

	if (ImGui::IsKeyDown('S')) {
		translate_vec.z += -1.0f;
	}

	if (ImGui::IsKeyDown('D')) {
		translate_vec.x += 1.0f;
	}

	if (ImGui::IsKeyDown(' ')) {
		translate_vec.y += -1.0f;
	}

	if (ImGui::IsKeyDown('C')) {
		translate_vec.y += 1.0f;
	}

	translate_vec *= m_camera_velocity * m_frame_ms.val() / 1000.0f;
	if (shift) {
		translate_vec *= 5.0f;
	}

	if (translate_vec != vec3(0.0f)) {
		m_fps_camera = true;

		// If VR is active, movement that isn't aligned with the current view
		// direction is _very_ jarring to the user, so make keyboard-based
		// movement aligned with the VR view, even though it is not an intended
		// movement mechanism. (Users should use controllers.)
		translate_camera(translate_vec, m_hmd && m_hmd->is_visible() ? mat3(m_views.front().camera0) : mat3(m_camera));
	}

	return false;
}

void Testbed::mouse_wheel() {
	float delta = ImGui::GetIO().MouseWheel;
	if (delta == 0) {
		return;
	}

	float scale_factor = pow(1.1f, -delta);
	set_scale(m_scale * scale_factor);

	reset_accumulation(true);
}

mat3 Testbed::rotation_from_angles(const vec2& angles) const {
	vec3 up = m_up_dir;
	vec3 side = m_camera[0];
	return rotmat(angles.x, up) * rotmat(angles.y, side);
}

void Testbed::mouse_drag() {
	vec2 rel = vec2{ImGui::GetIO().MouseDelta.x, ImGui::GetIO().MouseDelta.y} / (float)m_window_res[m_fov_axis];
	vec2 mouse = {ImGui::GetMousePos().x, ImGui::GetMousePos().y};

	vec3 side = m_camera[0];

	bool shift = ImGui::GetIO().KeyShift;

	// Left pressed
	if (ImGui::GetIO().MouseClicked[0] && shift) {
		m_autofocus_target = get_3d_pos_from_pixel(*m_views.front().render_buffer, mouse);
		m_autofocus = true;

		reset_accumulation();
	}

	// Left held
	if (ImGui::GetIO().MouseDown[0]) {
		float rot_sensitivity = m_fps_camera ? 0.35f : 1.0f;
		mat3 rot = rotation_from_angles(-rel * 2.0f * PI() * rot_sensitivity);

		if (m_fps_camera) {
			rot *= mat3(m_camera);
			m_camera = mat4x3(rot[0], rot[1], rot[2], m_camera[3]);
		} else {
			// Turntable
			auto old_look_at = look_at();
			set_look_at({0.0f, 0.0f, 0.0f});
			m_camera = rot * m_camera;
			set_look_at(old_look_at);
		}

		reset_accumulation(true);
	}

	// Right held
	if (ImGui::GetIO().MouseDown[1]) {
		mat3 rot = rotation_from_angles(-rel * 2.0f * PI());
		if (m_render_mode == ERenderMode::Shade) {
			m_sun_dir = transpose(rot) * m_sun_dir;
		}

		m_slice_plane_z += -rel.y * m_bounding_radius;
		reset_accumulation();
	}

	// Middle pressed
	if (ImGui::GetIO().MouseClicked[2]) {
		m_drag_depth = get_depth_from_renderbuffer(*m_views.front().render_buffer, mouse / vec2(m_window_res));
	}

	// Middle held
	if (ImGui::GetIO().MouseDown[2]) {
		vec3 translation = vec3{-rel.x, -rel.y, 0.0f} / m_zoom;
		bool is_orthographic = m_render_with_lens_distortion && m_render_lens.mode == ELensMode::Orthographic;

		translation /= m_relative_focal_length[m_fov_axis];

		// If we have a valid depth value, scale the scene translation by it such that the
		// hovered point in 3D space stays under the cursor.
		if (m_drag_depth < 256.0f && !is_orthographic) {
			translation *= m_drag_depth;
		}

		translate_camera(translation, mat3(m_camera));
	}
}

bool Testbed::begin_frame() {
	if (glfwWindowShouldClose(m_glfw_window)) {
		destroy_window();
		return false;
	}

	{
		auto now = std::chrono::steady_clock::now();
		auto elapsed = now - m_last_frame_time_point;
		m_last_frame_time_point = now;
		m_frame_ms.update(std::chrono::duration<float, std::milli>(elapsed).count());
	}

	glfwPollEvents();
	glfwGetFramebufferSize(m_glfw_window, &m_window_res.x, &m_window_res.y);

	ImGui_ImplOpenGL3_NewFrame();
	ImGui_ImplGlfw_NewFrame();
	ImGui::NewFrame();
	ImGuizmo::BeginFrame();

	return true;
}

void Testbed::handle_user_input() {
	// Only respond to mouse inputs when not interacting with ImGui
	if (!ImGui::IsAnyItemActive() && !ImGuizmo::IsUsing() && !ImGui::GetIO().WantCaptureMouse) {
		mouse_wheel();
		mouse_drag();
	}

	keyboard_event();

	switch (m_imgui.mode) {
		case ImGuiMode::Enabled: imgui(); break;
		case ImGuiMode::FpsOverlay: overlay_fps(); break;
		case ImGuiMode::Disabled: break;
		default: throw std::runtime_error{fmt::format("Invalid imgui mode: {}", (uint32_t)m_imgui.mode)};
	}
}

vec3 Testbed::vr_to_world(const vec3& pos) const { return mat3(m_camera) * pos * m_scale + m_camera[3]; }

void Testbed::begin_vr_frame_and_handle_vr_input() {
	if (!m_hmd) {
		m_vr_frame_info = nullptr;
		return;
	}

	m_hmd->poll_events();
	if (!m_hmd->must_run_frame_loop()) {
		m_vr_frame_info = nullptr;
		return;
	}

	m_vr_frame_info = m_hmd->begin_frame();

	const auto& views = m_vr_frame_info->views;
	size_t n_views = views.size();
	size_t n_devices = m_devices.size();
	if (n_views > 0) {
		set_n_views(n_views);

		ivec2 total_size = 0;
		for (size_t i = 0; i < n_views; ++i) {
			ivec2 view_resolution = {views[i].view.subImage.imageRect.extent.width, views[i].view.subImage.imageRect.extent.height};
			total_size += view_resolution;

			m_views[i].full_resolution = view_resolution;

			// Apply the VR pose relative to the world camera transform.
			m_views[i].camera0 = mat3(m_camera) * views[i].pose;
			m_views[i].camera0[3] = vr_to_world(views[i].pose[3]);
			m_views[i].camera1 = m_views[i].camera0;

			m_views[i].visualized_dimension = m_visualized_dimension;

			const auto& xr_fov = views[i].view.fov;

			// Compute the distance on the image plane (1 unit away from the camera) that an angle of the respective FOV spans
			vec2 rel_focal_length_left_down = 0.5f *
				fov_to_focal_length(ivec2(1), vec2{360.0f * xr_fov.angleLeft / PI(), 360.0f * xr_fov.angleDown / PI()});
			vec2 rel_focal_length_right_up = 0.5f *
				fov_to_focal_length(ivec2(1), vec2{360.0f * xr_fov.angleRight / PI(), 360.0f * xr_fov.angleUp / PI()});

			// Compute total distance (for X and Y) that is spanned on the image plane.
			m_views[i].relative_focal_length = rel_focal_length_right_up - rel_focal_length_left_down;

			// Compute fraction of that distance that is spanned by the right-up part and set screen center accordingly.
			vec2 ratio = rel_focal_length_right_up / m_views[i].relative_focal_length;
			m_views[i].screen_center = {1.0f - ratio.x, ratio.y};

			// Fix up weirdness in the rendering pipeline
			m_views[i].relative_focal_length[(m_fov_axis + 1) % 2] *= (float)view_resolution[(m_fov_axis + 1) % 2] /
				(float)view_resolution[m_fov_axis];
			m_views[i].render_buffer->set_hidden_area_mask(m_vr_use_hidden_area_mask ? views[i].hidden_area_mask : nullptr);

			// Render each view on a different GPU (if available)
			m_views[i].device = m_use_aux_devices ? &m_devices.at(i % m_devices.size()) : &primary_device();
		}

		// Put all the views next to each other, but at half size
		glfwSetWindowSize(m_glfw_window, total_size.x / 2, (total_size.y / 2) / n_views);

		// VR controller input
		const auto& hands = m_vr_frame_info->hands;
		m_fps_camera = true;

		// TRANSLATE BY STICK (if not pressing the stick)
		if (!hands[0].pressing) {
			vec3 translate_vec = vec3{hands[0].thumbstick.x, 0.0f, hands[0].thumbstick.y} * m_camera_velocity * m_frame_ms.val() / 1000.0f;
			if (translate_vec != vec3(0.0f)) {
				translate_camera(translate_vec, mat3(m_views.front().camera0), false);
			}
		}

		// TURN BY STICK (if not pressing the stick)
		if (!hands[1].pressing) {
			auto prev_camera = m_camera;

			// Turn around the up vector (equivalent to x-axis mouse drag) with right joystick left/right
			float sensitivity = 0.35f;
			auto rot = rotation_from_angles({-2.0f * PI() * sensitivity * hands[1].thumbstick.x * m_frame_ms.val() / 1000.0f, 0.0f}) *
				mat3(m_camera);
			m_camera = mat4x3(rot[0], rot[1], rot[2], m_camera[3]);

			// Translate camera such that center of rotation was about the current view
			m_camera[3] += mat3(prev_camera) * views[0].pose[3] * m_scale - mat3(m_camera) * views[0].pose[3] * m_scale;
		}

		// TRANSLATE, SCALE, AND ROTATE BY GRAB
		{
			bool both_grabbing = hands[0].grabbing && hands[1].grabbing;
			float drag_factor = both_grabbing ? 0.5f : 1.0f;

			if (both_grabbing) {
				drag_factor = 0.5f;

				vec3 prev_diff = hands[0].prev_grab_pos - hands[1].prev_grab_pos;
				vec3 diff = hands[0].grab_pos - hands[1].grab_pos;
				vec3 center = 0.5f * (hands[0].grab_pos + hands[1].grab_pos);

				vec3 center_world = vr_to_world(0.5f * (hands[0].grab_pos + hands[1].grab_pos));

				// Scale around center position of the two dragging hands. Makes the scaling feel similar to phone pinch-to-zoom
				float scale = m_scale * length(prev_diff) / length(diff);
				m_camera[3] = (view_pos() - center_world) * (scale / m_scale) + center_world;
				m_scale = scale;

				// Take rotational component and project it to the nearest rotation about the up vector.
				// We don't want to rotate the scene about any other axis.
				vec3 rot = cross(normalize(prev_diff), normalize(diff));
				float rot_radians = std::asin(dot(m_up_dir, rot));

				auto prev_camera = m_camera;
				auto rotcam = rotmat(rot_radians, m_up_dir) * mat3(m_camera);
				m_camera = mat4x3(rotcam[0], rotcam[1], rotcam[2], m_camera[3]);
				m_camera[3] += mat3(prev_camera) * center * m_scale - mat3(m_camera) * center * m_scale;
			}

			for (const auto& hand : hands) {
				if (hand.grabbing) {
					m_camera[3] -= drag_factor * mat3(m_camera) * hand.drag() * m_scale;
				}
			}
		}
	}
}

void Testbed::SecondWindow::draw(GLuint texture) {
	if (!window) {
		return;
	}
	int display_w, display_h;
	GLFWwindow* old_context = glfwGetCurrentContext();
	glfwMakeContextCurrent(window);
	glfwGetFramebufferSize(window, &display_w, &display_h);
	glViewport(0, 0, display_w, display_h);
	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glEnable(GL_TEXTURE_2D);
	glBindTexture(GL_TEXTURE_2D, texture);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glBindVertexArray(vao);
	if (program) {
		glUseProgram(program);
	}
	glDrawArrays(GL_TRIANGLES, 0, 6);
	glBindVertexArray(0);
	glUseProgram(0);
	glfwSwapBuffers(window);
	glfwMakeContextCurrent(old_context);
}

void Testbed::init_opengl_shaders() {
	static const char* shader_vert = R"glsl(#version 140
		out vec2 UVs;
		void main() {
			UVs = vec2((gl_VertexID << 1) & 2, gl_VertexID & 2);
			gl_Position = vec4(UVs * 2.0 - 1.0, 0.0, 1.0);
		})glsl";

	static const char* shader_frag = R"glsl(#version 140
		in vec2 UVs;
		out vec4 frag_color;
		uniform sampler2D rgba_texture;
		uniform sampler2D depth_texture;

		struct FoveationWarp {
			float al, bl, cl;
			float am, bm;
			float ar, br, cr;
			float switch_left, switch_right;
			float inv_switch_left, inv_switch_right;
		};

		uniform FoveationWarp warp_x;
		uniform FoveationWarp warp_y;

		float unwarp(in FoveationWarp warp, float y) {
			y = clamp(y, 0.0, 1.0);
			if (y < warp.inv_switch_left) {
				return (sqrt(-4.0 * warp.al * warp.cl + 4.0 * warp.al * y + warp.bl * warp.bl) - warp.bl) / (2.0 * warp.al);
			} else if (y > warp.inv_switch_right) {
				return (sqrt(-4.0 * warp.ar * warp.cr + 4.0 * warp.ar * y + warp.br * warp.br) - warp.br) / (2.0 * warp.ar);
			} else {
				return (y - warp.bm) / warp.am;
			}
		}

		vec2 unwarp(in vec2 pos) {
			return vec2(unwarp(warp_x, pos.x), unwarp(warp_y, pos.y));
		}

		void main() {
			vec2 tex_coords = UVs;
			tex_coords.y = 1.0 - tex_coords.y;
			tex_coords = unwarp(tex_coords);
			frag_color = texture(rgba_texture, tex_coords.xy);
			//Uncomment the following line of code to visualize debug the depth buffer for debugging.
			// frag_color = vec4(vec3(texture(depth_texture, tex_coords.xy).r), 1.0);
			gl_FragDepth = texture(depth_texture, tex_coords.xy).r;
		})glsl";

	GLuint vert = glCreateShader(GL_VERTEX_SHADER);
	glShaderSource(vert, 1, &shader_vert, NULL);
	glCompileShader(vert);
	check_shader(vert, "Blit vertex shader", false);

	GLuint frag = glCreateShader(GL_FRAGMENT_SHADER);
	glShaderSource(frag, 1, &shader_frag, NULL);
	glCompileShader(frag);
	check_shader(frag, "Blit fragment shader", false);

	m_blit_program = glCreateProgram();
	glAttachShader(m_blit_program, vert);
	glAttachShader(m_blit_program, frag);
	glLinkProgram(m_blit_program);
	check_shader(m_blit_program, "Blit shader program", true);

	glDeleteShader(vert);
	glDeleteShader(frag);

	glGenVertexArrays(1, &m_blit_vao);
}

void Testbed::blit_texture(
	const Foveation& foveation,
	GLint rgba_texture,
	GLint rgba_filter_mode,
	GLint depth_texture,
	GLint framebuffer,
	const ivec2& offset,
	const ivec2& resolution
) {
	if (m_blit_program == 0) {
		return;
	}

	// Blit image to OpenXR swapchain.
	// Note that the OpenXR swapchain is 8bit while the rendering is in a float texture.
	// As some XR runtimes do not support float swapchains, we can't render into it directly.

	bool tex = glIsEnabled(GL_TEXTURE_2D);
	bool depth = glIsEnabled(GL_DEPTH_TEST);
	bool cull = glIsEnabled(GL_CULL_FACE);

	if (!tex) {
		glEnable(GL_TEXTURE_2D);
	}
	if (!depth) {
		glEnable(GL_DEPTH_TEST);
	}
	if (cull) {
		glDisable(GL_CULL_FACE);
	}

	glDepthFunc(GL_ALWAYS);
	glDepthMask(GL_TRUE);

	glBindVertexArray(m_blit_vao);
	glUseProgram(m_blit_program);
	glUniform1i(glGetUniformLocation(m_blit_program, "rgba_texture"), 0);
	glUniform1i(glGetUniformLocation(m_blit_program, "depth_texture"), 1);

	auto bind_warp = [&](const FoveationPiecewiseQuadratic& warp, const std::string& uniform_name) {
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".al").c_str()), warp.al);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".bl").c_str()), warp.bl);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".cl").c_str()), warp.cl);

		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".am").c_str()), warp.am);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".bm").c_str()), warp.bm);

		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".ar").c_str()), warp.ar);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".br").c_str()), warp.br);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".cr").c_str()), warp.cr);

		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".switch_left").c_str()), warp.switch_left);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".switch_right").c_str()), warp.switch_right);

		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".inv_switch_left").c_str()), warp.inv_switch_left);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".inv_switch_right").c_str()), warp.inv_switch_right);
	};

	bind_warp(foveation.warp_x, "warp_x");
	bind_warp(foveation.warp_y, "warp_y");

	glActiveTexture(GL_TEXTURE1);
	glBindTexture(GL_TEXTURE_2D, depth_texture);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_2D, rgba_texture);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, rgba_filter_mode);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, rgba_filter_mode);

	glBindFramebuffer(GL_FRAMEBUFFER, framebuffer);
	glViewport(offset.x, offset.y, resolution.x, resolution.y);

	glDrawArrays(GL_TRIANGLES, 0, 3);

	glBindVertexArray(0);
	glUseProgram(0);

	glDepthFunc(GL_LESS);

	// restore old state
	if (!tex) {
		glDisable(GL_TEXTURE_2D);
	}
	if (!depth) {
		glDisable(GL_DEPTH_TEST);
	}
	if (cull) {
		glEnable(GL_CULL_FACE);
	}
	glBindFramebuffer(GL_FRAMEBUFFER, 0);
}

void Testbed::draw_gui() {
	// Make sure all the cuda code finished its business here
	CUDA_CHECK_THROW(hipDeviceSynchronize());

	if (!m_rgba_render_textures.empty()) {
		m_second_window.draw((GLuint)m_rgba_render_textures.front()->texture());
	}

	glfwMakeContextCurrent(m_glfw_window);
	int display_w, display_h;
	glfwGetFramebufferSize(m_glfw_window, &display_w, &display_h);
	glViewport(0, 0, display_w, display_h);
	glClearColor(0.f, 0.f, 0.f, 0.f);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glEnable(GL_BLEND);
	glBlendEquationSeparate(GL_FUNC_ADD, GL_FUNC_ADD);
	glBlendFuncSeparate(GL_ONE, GL_ONE_MINUS_SRC_ALPHA, GL_ONE, GL_ONE_MINUS_SRC_ALPHA);

	ivec2 extent = {(int)((float)display_w / m_n_views.x), (int)((float)display_h / m_n_views.y)};

	int i = 0;
	for (int y = 0; y < m_n_views.y; ++y) {
		for (int x = 0; x < m_n_views.x; ++x) {
			if (i >= m_views.size()) {
				break;
			}

			auto& view = m_views[i];
			ivec2 top_left{x * extent.x, display_h - (y + 1) * extent.y};
			blit_texture(
				m_foveated_rendering_visualize ? Foveation{} : view.foveation,
				m_rgba_render_textures.at(i)->texture(),
				m_foveated_rendering ? GL_LINEAR : GL_NEAREST,
				m_depth_render_textures.at(i)->texture(),
				0,
				top_left,
				extent
			);

			++i;
		}
	}
	glFinish();
	glViewport(0, 0, display_w, display_h);

	ImDrawList* list = ImGui::GetBackgroundDrawList();
	list->AddCallback(ImDrawCallback_ResetRenderState, nullptr);

	// Visualizations are only meaningful when rendering a single view
	if (m_views.size() == 1) {
		draw_visualizations(list, m_smoothed_camera);
	}

	if (m_render_ground_truth) {
		list->AddText(ImVec2(4.f, 4.f), 0xffffffff, "Ground Truth");
	}

	ImGui::Render();
	ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

	glfwSwapBuffers(m_glfw_window);

	// Make sure all the OGL code finished its business here.
	// Any code outside of this function needs to be able to freely write to
	// textures without being worried about interfering with rendering.
	glFinish();
}
#endif // NGP_GUI

__global__ void to_8bit_color_kernel(ivec2 resolution, EColorSpace output_color_space, hipSurfaceObject_t surface, uint8_t* result) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= resolution.x || y >= resolution.y) {
		return;
	}

	vec4 color;
	surf2Dread((float4*)&color, surface, x * sizeof(float4), y);

	if (output_color_space == EColorSpace::Linear) {
		color.rgb() = linear_to_srgb(color.rgb());
	}

	for (uint32_t i = 0; i < 3; ++i) {
		result[(x + resolution.x * y) * 3 + i] = (uint8_t)(clamp(color[i], 0.0f, 1.0f) * 255.0f + 0.5f);
	}
}

void Testbed::prepare_next_camera_path_frame() {
	if (!m_camera_path.rendering) {
		return;
	}

	// If we're rendering a video, we'd like to accumulate multiple spp
	// for motion blur. Hence dump the frame once the target spp has been reached
	// and only reset _then_.
	if (m_views.front().render_buffer->spp() == m_camera_path.render_settings.spp) {
		auto tmp_dir = fs::path{"tmp"};
		if (!tmp_dir.exists()) {
			if (!fs::create_directory(tmp_dir)) {
				m_camera_path.rendering = false;
				tlog::error() << "Failed to create temporary directory 'tmp' to hold rendered images.";
				return;
			}
		}

		ivec2 res = m_views.front().render_buffer->out_resolution();
		const dim3 threads = {16, 8, 1};
		const dim3 blocks = {div_round_up((uint32_t)res.x, threads.x), div_round_up((uint32_t)res.y, threads.y), 1};

		GPUMemory<uint8_t> image_data(product(res) * 3);
		to_8bit_color_kernel<<<blocks, threads>>>(
			res,
			EColorSpace::SRGB, // the GUI always renders in SRGB
			m_views.front().render_buffer->surface(),
			image_data.data()
		);

		m_render_futures.emplace_back(
			m_thread_pool.enqueue_task([image_data = std::move(image_data), frame_idx = m_camera_path.render_frame_idx++, res, tmp_dir] {
				std::vector<uint8_t> cpu_image_data(image_data.size());
				CUDA_CHECK_THROW(hipMemcpy(cpu_image_data.data(), image_data.data(), image_data.bytes(), hipMemcpyDeviceToHost));
				write_stbi(tmp_dir / fmt::format("{:06d}.jpg", frame_idx), res.x, res.y, 3, cpu_image_data.data(), 100);
			})
		);

		reset_accumulation(true);

		if (m_camera_path.render_frame_idx == m_camera_path.render_settings.n_frames(m_camera_path.duration_seconds())) {
			m_camera_path.rendering = false;

			wait_all(m_render_futures);
			m_render_futures.clear();

			tlog::success() << "Finished rendering '.jpg' video frames to '" << tmp_dir << "'. Assembling them into a video next.";

			fs::path ffmpeg = "ffmpeg";

#ifdef _WIN32
			// Under Windows, try automatically downloading FFmpeg binaries if they don't exist
			if (system(fmt::format("where {} >nul 2>nul", ffmpeg.str()).c_str()) != 0) {
				fs::path dir = root_dir();
				if ((dir / "external" / "ffmpeg").exists()) {
					for (const auto& path : fs::directory{dir / "external" / "ffmpeg"}) {
						ffmpeg = path / "bin" / "ffmpeg.exe";
					}
				}

				if (!ffmpeg.exists()) {
					tlog::info() << "FFmpeg not found. Downloading FFmpeg...";
					do_system((dir / "scripts" / "download_ffmpeg.bat").str());
				}

				for (const auto& path : fs::directory{dir / "external" / "ffmpeg"}) {
					ffmpeg = path / "bin" / "ffmpeg.exe";
				}

				if (!ffmpeg.exists()) {
					tlog::warning() << "FFmpeg download failed. Trying system-wide FFmpeg.";
				}
			}
#endif

			auto ffmpeg_command = fmt::format(
				"{} -loglevel error -y -framerate {} -i tmp/%06d.jpg -c:v libx264 -preset slow -crf {} -pix_fmt yuv420p \"{}\"",
				ffmpeg.str(),
				m_camera_path.render_settings.fps,
				// Quality goes from 0 to 10. This conversion to CRF means a quality of 10
				// is a CRF of 17 and a quality of 0 a CRF of 27, which covers the "sane"
				// range of x264 quality settings according to the FFmpeg docs:
				// https://trac.ffmpeg.org/wiki/Encode/H.264
				27 - m_camera_path.render_settings.quality,
				m_camera_path.render_settings.filename
			);
			int ffmpeg_result = do_system(ffmpeg_command);
			if (ffmpeg_result == 0) {
				tlog::success() << "Saved video '" << m_camera_path.render_settings.filename << "'";
			} else if (ffmpeg_result == -1) {
				tlog::error() << "Video could not be assembled: FFmpeg not found.";
			} else {
				tlog::error() << "Video could not be assembled: FFmpeg failed";
			}

			clear_tmp_dir();
		}
	}

	const auto& rs = m_camera_path.render_settings;
	const float duration = m_camera_path.duration_seconds();
	m_camera_path.play_time = (float)((double)m_camera_path.render_frame_idx / (double)rs.n_frames(duration));

	if (m_views.front().render_buffer->spp() == 0) {
		set_camera_from_time(m_camera_path.play_time);
		apply_camera_smoothing(rs.frame_milliseconds(duration));

		auto smoothed_camera_backup = m_smoothed_camera;

		// Compute the camera for the next frame in order to be able to compute motion blur
		// between it and the current one.
		set_camera_from_time(m_camera_path.play_time + 1.0f / rs.n_frames(duration));
		apply_camera_smoothing(rs.frame_milliseconds(duration));

		m_camera_path.render_frame_end_camera = m_smoothed_camera;

		// Revert camera such that the next frame will be computed correctly
		// (Start camera of next frame should be the same as end camera of this frame)
		set_camera_from_time(m_camera_path.play_time);
		m_smoothed_camera = smoothed_camera_backup;
	}
}

__global__ void reproject_kernel(
	BoundingBox render_aabb,
	mat3 render_aabb_to_local,
	default_rng_t rng,
	float near_t,
	float step_factor,
	uint32_t spp,
	uint32_t view_idx,
	mat4x3 src_camera,
	vec2 src_screen_center,
	vec2 src_focal_length,
	ivec2 src_resolution,
	Foveation src_foveation,
	Lens src_lens,
	MatrixView<const float> src_depth_buffer,
	mat4x3 dst_camera,
	vec2 dst_screen_center,
	vec2 dst_focal_length,
	ivec2 dst_resolution,
	Foveation dst_foveation,
	Lens dst_lens,
	vec4* __restrict__ dst_frame_buffer,
	MatrixView<float> dst_depth_buffer,
	MatrixView<uint8_t> dst_hole_mask,
	MatrixView<ViewIdx> dst_index_field,
	MatrixView<uint8_t> src_hole_mask = {},
	MatrixView<ViewIdx> src_index_field = {}
) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	uint32_t is_hole = dst_hole_mask(y, x);
	if (x >= dst_resolution.x || y >= dst_resolution.y || (src_hole_mask && !is_hole)) {
		return;
	}

	auto ray = pixel_to_ray(
		spp,
		{(int)x, (int)y},
		dst_resolution,
		dst_focal_length,
		dst_camera,
		dst_screen_center,
		vec3(0.0f), // parallax
		false,      // pixel center snap
		0.0f,       // near dist
		1.0f,       // focus
		0.0f,       // aperture
		dst_foveation,
		{},
		dst_lens
	);

	uint32_t dst_idx = x + dst_resolution.x * y;

	float t = near_t;
	rng.advance(dst_idx);
	t *= std::pow(step_factor, rng.next_float());

	struct Result {
		ViewIdx idx;
		float dist;
		float t;
	};

	auto get_reprojected_dist = [&](float t) -> Result {
		vec3 p = ray(t);

		vec2 src_px = pos_to_pixel(p, src_resolution, src_focal_length, src_camera, src_screen_center, vec3(0.0f), src_foveation, src_lens);

		if (src_px.x <= 0 || src_px.x >= src_resolution.x || src_px.y <= 0 || src_px.y >= src_resolution.y) {
			return {
				{-1, 0},
                -1.0f, -1.0f
			};
		}

		ViewIdx nearest = {clamp(ivec2(floor(src_px)), 0, src_resolution - 1), view_idx};
		if (src_hole_mask) {
			if (!src_hole_mask(nearest.px.y, nearest.px.x) || src_depth_buffer(nearest.px.y, nearest.px.x) == 0.0f) {
				return {
					{-1, 0},
                    -1.0f, -1.0f
				};
			}
		}

		float d = src_depth_buffer(nearest.px.y, nearest.px.x);
		Ray src_ray = {
			src_camera[3],
			p - src_camera[3],
		};

		src_ray.d /= src_lens.is_360() ? length(src_ray.d) : dot(src_ray.d, src_camera[2]);

		vec3 src_p = src_ray(d);
		if (src_index_field) {
			nearest = src_index_field(nearest.px.y, nearest.px.x);
		}

		return {nearest, distance(p, src_p), t};
	};

	auto refine_match = [&](Result match) -> Result {
		static const uint32_t N_STEPS_PER_REFINEMENT = 10;
		static const uint32_t N_REFINEMENTS = 3;

		float prev_t = match.t / step_factor;
		float next_t = match.t * step_factor;

		NGP_PRAGMA_UNROLL
		for (uint32_t j = 0; j < N_REFINEMENTS; ++j) {
			float step_size = (next_t - prev_t) / (N_STEPS_PER_REFINEMENT - 1);
			float t = prev_t;

			NGP_PRAGMA_UNROLL
			for (uint32_t i = 0; i < N_STEPS_PER_REFINEMENT; ++i) {
				auto res = get_reprojected_dist(t);
				if (res.idx.px.x >= 0 && res.dist < match.dist) {
					match = res;
					prev_t = t - step_size;
					next_t = t + step_size;
				}

				t += step_size;
			}
		}

		return match;
	};

	Result final = {
		{-1, 0},
        std::numeric_limits<float>::infinity(), 0
	};
	Result fallback = final;

	float mint = fmaxf(render_aabb.ray_intersect(render_aabb_to_local * ray.o, render_aabb_to_local * ray.d).x, 0.0f) + 1e-6f;
	if (mint < MAX_DEPTH()) {
		while (t <= mint) {
			t *= step_factor;
		}
	}

	// float last_dist = std::numeric_limits<float>::infinity();
	for (; render_aabb.contains(render_aabb_to_local * ray(t)); t *= step_factor) {
		auto res = get_reprojected_dist(t);
		if (res.idx.px.x >= 0) {
			if (res.dist < t * (step_factor - 1.0f)) {
				res = refine_match(res);
				if (res.dist < final.dist) {
					if (res.dist / res.t < 4.0f / dst_focal_length.x) {
						final = res;
						break;
					}
				}
			}

			// if (res.dist < last_dist) {
			//	fallback = res;
			// }

			// last_dist = res.dist;
		}
	}

	if (final.idx.px.x == -1) {
		final = fallback;
	}

	float prev_depth = dst_depth_buffer(y, x);

	dst_frame_buffer[dst_idx] = vec4::zero();
	if (final.idx.px.x == -1) {
		if (is_hole) {
			dst_depth_buffer(y, x) = MAX_DEPTH();
			dst_hole_mask(y, x) = 1;
			dst_index_field(y, x) = {-1, 0};
		}
	} else {
		if (is_hole || final.t * step_factor < prev_depth) {
			dst_depth_buffer(y, x) = final.t;
			dst_hole_mask(y, x) = src_index_field ? 2 : 0;
			dst_index_field(y, x) = final.idx;
		}
	}
}

__global__ void dilate_holes_kernel(ivec2 res, MatrixView<const uint8_t> old_hole_mask, MatrixView<uint8_t> hole_mask) {
	int32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	int32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= res.x || y >= res.y) {
		return;
	}

	auto is_hole = [&](const ivec2& offset) {
		auto clamped = clamp(ivec2{x, y} + offset, 0, res - 1);
		return old_hole_mask(clamped.y, clamped.x);
	};

	hole_mask(y, x) = is_hole({1, 0}) || is_hole({-1, 0}) || is_hole({1, 1}) || is_hole({-1, 1}) || is_hole({1, -1}) || is_hole({-1, -1}) ||
		is_hole({0, 1}) || is_hole({0, -1});
}

__global__ void generate_alt_depth_kernel(
	mat4x3 src_camera,
	vec2 src_screen_center,
	vec2 src_focal_length,
	ivec2 src_resolution,
	const vec4* __restrict__ src_frame_buffer,
	const float* __restrict__ src_depth_buffer,
	Foveation src_foveation,
	Lens src_lens,
	mat4x3 dst_camera,
	Lens dst_lens,
	MatrixView<float> alt_depth_buffer
) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= src_resolution.x || y >= src_resolution.y) {
		return;
	}

	auto ray = pixel_to_ray(
		0,
		{(int)x, (int)y},
		src_resolution,
		src_focal_length,
		src_camera,
		src_screen_center,
		vec3(0.0f), // parallax
		false,      // pixel center snap
		0.0f,       // near dist
		1.0f,       // focus
		0.0f,       // aperture
		src_foveation,
		{},
		src_lens
	);

	uint32_t src_idx = x + src_resolution.x * y;
	vec3 p = ray(src_depth_buffer[src_idx]);

	alt_depth_buffer(y, x) = dst_lens.is_360() ? distance(p, dst_camera[3]) : dot(p - dst_camera[3], dst_camera[2]);
}

__global__ void copy_depth_buffer_kernel(ivec2 dst_resolution, const float* __restrict__ src_depth_buffer, MatrixView<float> dst_depth_buffer) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= dst_resolution.x || y >= dst_resolution.y) {
		return;
	}

	uint32_t idx = x + dst_resolution.x * y;
	dst_depth_buffer(y, x) = src_depth_buffer[idx];
}

static constexpr float Z_NEAR = 0.1f;
static constexpr float Z_BASE = 1.03f;

inline NGP_HOST_DEVICE float to_log_depth(float d) { return logf(d / Z_NEAR) * logf(Z_BASE); }

inline NGP_HOST_DEVICE float from_log_depth(float d) { return expf(d / logf(Z_BASE)) * Z_NEAR; }

inline NGP_HOST_DEVICE vec4 from_rgbd32(uint32_t val) {
	vec4 result = rgba32_to_rgba(val);
	result.a = from_log_depth(result.a);
	return result;
}

inline NGP_HOST_DEVICE uint32_t to_rgbd32(vec4 rgbd) {
	rgbd.a = to_log_depth(rgbd.a);
	return rgba_to_rgba32(rgbd);
}

__global__ void reproject_viz_kernel(
	ivec2 dst_res,
	const ivec2* src_res,
	bool pm_enable,
	MatrixView<const uint32_t> hole_labels,
	MatrixView<const EPmPixelState> state,
	MatrixView<const ViewIdx> index_field,
	MatrixView<const uint32_t> dst_rgbd,
	MatrixView<const float> dst_depth,
	const MatrixView<const uint32_t>* src_rgba,
	const MatrixView<const float>* src_depth,
	MatrixView<vec4> frame,
	MatrixView<float> depth,
	EPmVizMode viz_mode,
	float depth_scale
) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= dst_res.x || y >= dst_res.y) {
		return;
	}

	if (!pm_enable && state(y, x) == EPmPixelState::Hole) {
		if (viz_mode == EPmVizMode::Depth) {
			frame(y, x).rgb() = vec3(depth(y, x) * depth_scale);
		} else {
			frame(y, x).rgb() = vec3(0.0f);
		}

		depth(y, x) = MAX_DEPTH();
		return;
	}

	auto src_idx = index_field(y, x);

	if (viz_mode == EPmVizMode::Depth) {
		frame(y, x).rgb() = vec3(dst_depth(y, x) * depth_scale);
	} else if (viz_mode == EPmVizMode::Offset) {
		vec2 diff = vec2(x, y) / vec2(dst_res) - vec2(src_idx.px) / vec2(src_res[src_idx.view]);
		float l = length(diff);
		frame(y, x).rgb() = hsv_to_rgb({atan2(diff.y / l, diff.x / l) / (PI() * 2.0f) + 0.5f, 1.0f, l});
	} else if (viz_mode == EPmVizMode::Holes) {
		if (state(y, x) == EPmPixelState::Hole) {
			frame(y, x).rgb() = colormap_turbo(hole_labels(y, x) / (float)product(dst_res));
		}
	} else {
		vec4 rgbd = rgba32_to_rgba(src_rgba[src_idx.view](src_idx.px.y, src_idx.px.x));
		rgbd.rgb() = srgb_to_linear(rgbd.rgb());
		frame(y, x) = rgbd;
		depth(y, x) = src_depth[src_idx.view](src_idx.px.y, src_idx.px.x);
	}
}

static constexpr int32_t PM_PATCH_RADIUS = 4;

inline NGP_HOST_DEVICE ivec2 mirror(const ivec2& v, const ivec2& res) { return abs(res - abs(res - v - 1) - 1); }

__global__ void pm_prepare_padded_src_buffers(
	ivec2 padded_res,
	ivec2 res,
	MatrixView<const vec4> src_rgba,
	MatrixView<const float> src_depth,
	MatrixView<uint32_t> dst_rgbd,
	MatrixView<float> dst_depth
) {
	int32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	int32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= padded_res.x || y >= padded_res.y) {
		return;
	}

	ivec2 padding = (padded_res - res) / 2;
	ivec2 idx = {(int16_t)(x - padding.x), (int16_t)(y - padding.y)};

	// auto clamped_idx = clamp(idx, i16vec2((int16_t)0), i16vec2(res - 1));
	auto clamped_idx = mirror(idx, i16vec2(res));

	vec4 rgba = src_rgba(clamped_idx.y, clamped_idx.x);
	rgba.rgb() = linear_to_srgb(rgba.rgb());
	dst_rgbd(idx.y, idx.x) = rgba_to_rgba32(rgba);
	dst_depth(idx.y, idx.x) = src_depth(clamped_idx.y, clamped_idx.x);
}

__global__ void pm_prepare_padded_dst_buffers(
	ivec2 padded_dst_res,
	ivec2 dst_res,
	uint32_t n_src_views,
	const ivec2* src_res,
	default_rng_t fixed_seed_rng,
	const MatrixView<const uint32_t>* src_rgbd,
	const MatrixView<const float>* src_depth,
	MatrixView<EPmPixelState> dst_state,
	MatrixView<ViewIdx> dst_index_field,
	MatrixView<uint32_t> dst_rgbd,
	MatrixView<float> dst_depth,
	MatrixView<float> dst_depth_threshold,
	MatrixView<const uint8_t> hole_mask
) {
	int32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	int32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= padded_dst_res.x || y >= padded_dst_res.y) {
		return;
	}

	ivec2 padding = (padded_dst_res - dst_res) / 2;
	ivec2 idx = {x - padding.x, y - padding.y};

	// auto clamped_idx = clamp(idx, i16vec2((int16_t)0), i16vec2(res - 1));
	auto clamped_idx = mirror(idx, dst_res);

	ViewIdx src_idx;
	uint8_t is_hole = hole_mask(clamped_idx.y, clamped_idx.x);
	if (is_hole == 1) {
		fixed_seed_rng.advance((x + y * padded_dst_res.x) * 3);

		// uint32_t random_view = fixed_seed_rng.next_uint(n_src_views);
		uint32_t random_view = 0;
		auto res = src_res[random_view];
		src_idx = {
			i16vec2{(int16_t)fixed_seed_rng.next_uint(res.y), (int16_t)fixed_seed_rng.next_uint(res.x)},
            random_view
		};
	} else {
		src_idx = dst_index_field(clamped_idx.y, clamped_idx.x);
	}

	dst_index_field(idx.y, idx.x) = src_idx;

	if (is_hole == 0) {
		dst_state(idx.y, idx.x) = EPmPixelState::Reprojected;
		dst_rgbd(idx.y, idx.x) = src_rgbd[src_idx.view](src_idx.px.y, src_idx.px.x);

		float depth = src_depth[src_idx.view](src_idx.px.y, src_idx.px.x);
		dst_depth(idx.y, idx.x) = depth;
		dst_depth_threshold(idx.y, idx.x) = depth;
	} else if (is_hole == 1) {
		dst_state(idx.y, idx.x) = EPmPixelState::Hole;
		dst_rgbd(idx.y, idx.x) = 0x00FF00FF;
		dst_depth(idx.y, idx.x) = 0.0f;
		dst_depth_threshold(idx.y, idx.x) = 0.0f;
	} else {
		dst_state(idx.y, idx.x) = EPmPixelState::Reprojected;
		dst_rgbd(idx.y, idx.x) = src_rgbd[src_idx.view](src_idx.px.y, src_idx.px.x);
		dst_depth_threshold(idx.y, idx.x) = dst_depth(idx.y, idx.x);
	}
}


void Testbed::reproject_views(const std::vector<const View*> src_views, View& dst_view) {
	if (src_views.empty()) {
		dst_view.render_buffer->clear_frame(m_stream.get());
		return;
	}

	auto dst_res = dst_view.render_buffer->in_resolution();

	std::vector<ivec2> src_res(src_views.size());
	std::vector<vec2> src_screen_center(src_views.size());
	std::vector<vec2> src_focal_length(src_views.size());
	std::vector<GPUImage<float>> tmp_src_depth_buffer(src_views.size());

	for (size_t i = 0; i < src_views.size(); ++i) {
		src_res[i] = src_views[i]->render_buffer->in_resolution();

		src_screen_center[i] = render_screen_center(src_views[i]->screen_center);
		src_focal_length[i] =
			calc_focal_length(src_views[i]->render_buffer->in_resolution(), src_views[i]->relative_focal_length, m_fov_axis, m_zoom);

		// Compute the depth of every pixel in the src_view when reprojected into the dst_view.
		// This could in principle happen in parallel with the reprojection step happening below.
		tmp_src_depth_buffer[i] = GPUImage<float>(src_res[i], m_stream.get());

		const dim3 threads = {16, 8, 1};
		const dim3 blocks = {div_round_up((uint32_t)dst_res.x, threads.x), div_round_up((uint32_t)dst_res.y, threads.y), 1};

		generate_alt_depth_kernel<<<blocks, threads, 0, m_stream.get()>>>(
			src_views[i]->camera0,
			src_screen_center[i],
			src_focal_length[i],
			src_res[i],
			src_views[i]->render_buffer->frame_buffer(),
			src_views[i]->render_buffer->depth_buffer(),
			src_views[i]->foveation,
			src_views[i]->lens,
			dst_view.camera0,
			dst_view.lens,
			tmp_src_depth_buffer[i].view()
		);
	}

	dst_view.render_buffer->clear_frame(m_stream.get());

	const dim3 threads = {16, 8, 1};
	const dim3 blocks = {div_round_up((uint32_t)dst_res.x, threads.x), div_round_up((uint32_t)dst_res.y, threads.y), 1};

	auto prev_index_field = std::move(dst_view.index_field);
	dst_view.index_field = GPUImage<ViewIdx>(dst_res, PM_PATCH_RADIUS, m_stream.get());

	auto prev_hole_mask = std::move(dst_view.hole_mask);
	dst_view.hole_mask = GPUImage<uint8_t>(dst_res, m_stream.get());
	dst_view.hole_mask.image.memset_async(m_stream.get(), 1);

	auto prev_depth_buffer = std::move(dst_view.depth_buffer);
	dst_view.depth_buffer = GPUImage<float>(dst_res, PM_PATCH_RADIUS, m_stream.get());

	auto dst_screen_center = render_screen_center(dst_view.screen_center);
	auto dst_focal_length = calc_focal_length(dst_res, dst_view.relative_focal_length, m_fov_axis, m_zoom);

	// First reproject from the source images as much as possible
	for (size_t i = 0; i < src_views.size(); ++i) {
		reproject_kernel<<<blocks, threads, 0, m_stream.get()>>>(
			m_render_aabb,
			m_render_aabb_to_local,
			m_rng,
			m_reproject_min_t,
			m_reproject_step_factor,
			dst_view.render_buffer->spp(),
			i,
			src_views[i]->camera0,
			src_screen_center[i],
			src_focal_length[i],
			src_res[i],
			src_views[i]->foveation,
			src_views[i]->lens,
			MatrixView<const float>(src_views[i]->render_buffer->depth_buffer(), src_res[i].x, 1),
			dst_view.camera0,
			dst_screen_center,
			dst_focal_length,
			dst_res,
			dst_view.foveation,
			dst_view.lens,
			dst_view.render_buffer->frame_buffer(),
			dst_view.depth_buffer.view(),
			dst_view.hole_mask.view(),
			dst_view.index_field.view()
		);
	}

	// auto old_holes_mask = std::move(dst_view.hole_mask);
	// dst_view.hole_mask = GPUImage<uint8_t>(dst_res, m_stream.get());
	// dilate_holes_kernel<<<blocks, threads, 0, m_stream.get()>>>(dst_res, old_holes_mask.view(), dst_view.hole_mask.view());

	// Then try reprojecting into the remaining holes from the previous rendering
	if (m_reproject_reuse_last_frame && prev_depth_buffer.data()) {
		reproject_kernel<<<blocks, threads, 0, m_stream.get()>>>(
			m_render_aabb,
			m_render_aabb_to_local,
			m_rng,
			m_reproject_min_t,
			m_reproject_step_factor,
			dst_view.render_buffer->spp(),
			0, // Reprojecting from the most recent view will copy the previous index anyway.
			dst_view.prev_camera,
			render_screen_center(dst_view.screen_center),
			calc_focal_length(prev_hole_mask.resolution(), dst_view.relative_focal_length, m_fov_axis, m_zoom),
			prev_hole_mask.resolution(),
			dst_view.prev_foveation,
			dst_view.lens,
			prev_depth_buffer.view(),
			dst_view.camera0,
			dst_screen_center,
			dst_focal_length,
			dst_res,
			dst_view.foveation,
			dst_view.lens,
			dst_view.render_buffer->frame_buffer(),
			dst_view.depth_buffer.view(),
			dst_view.hole_mask.view(),
			dst_view.index_field.view(),
			prev_hole_mask.view(),
			prev_index_field.view()
		);
	}

	m_rng.advance();

	auto hole_labels = GPUImage<uint32_t>(dst_res, m_stream.get());

	// Detect holes and label them
	{
		init_labels<<<blocks, threads, 0, m_stream.get()>>>(
			dst_res.x, dst_res.y, hole_labels.n_elements(), hole_labels.data(), dst_view.hole_mask.data()
		);
		resolve_labels<<<blocks, threads, 0, m_stream.get()>>>(dst_res.x, dst_res.y, hole_labels.n_elements(), hole_labels.data());
		label_reduction<<<blocks, threads, 0, m_stream.get()>>>(
			dst_res.x, dst_res.y, hole_labels.n_elements(), hole_labels.data(), dst_view.hole_mask.data()
		);
		resolve_labels<<<blocks, threads, 0, m_stream.get()>>>(dst_res.x, dst_res.y, hole_labels.n_elements(), hole_labels.data());
	}

	auto dst_state_buffer = GPUImage<EPmPixelState>(dst_res, PM_PATCH_RADIUS, m_stream.get());

	std::vector<GPUImage<uint32_t>> src_rgbd_buffer(src_views.size());
	std::vector<GPUImage<float>> src_depth_buffer(src_views.size());
	std::vector<ivec2> padded_src_res(src_views.size());

	std::vector<MatrixView<const uint32_t>> src_rgbd_views(src_views.size());
	std::vector<MatrixView<const float>> src_depth_views(src_views.size());

	for (size_t i = 0; i < src_views.size(); ++i) {
		src_rgbd_buffer[i] = GPUImage<uint32_t>(src_res[i], PM_PATCH_RADIUS, m_stream.get());
		src_depth_buffer[i] = GPUImage<float>(src_res[i], PM_PATCH_RADIUS, m_stream.get());
		padded_src_res[i] = src_rgbd_buffer[i].resolution_padded();

		const dim3 threads = {16, 8, 1};
		const dim3 blocks = {div_round_up((uint32_t)padded_src_res[i].x, threads.x), div_round_up((uint32_t)padded_src_res[i].y, threads.y), 1};

		pm_prepare_padded_src_buffers<<<blocks, threads, 0, m_stream.get()>>>(
			padded_src_res[i],
			src_res[i],
			MatrixView<const vec4>(src_views[i]->render_buffer->frame_buffer(), src_res[i].x, 1),
			tmp_src_depth_buffer[i].view(),
			src_rgbd_buffer[i].view(),
			src_depth_buffer[i].view()
		);

		src_rgbd_views[i] = src_rgbd_buffer[i].view();
		src_depth_views[i] = src_depth_buffer[i].view();
	}

	GPUMemoryArena::Allocation views_alloc;
	auto views_scratch = allocate_workspace_and_distribute<MatrixView<const uint32_t>, MatrixView<const float>, ivec2>(
		m_stream.get(), &views_alloc, src_views.size(), src_views.size(), src_views.size()
	);

	auto* src_rgba_views_device = std::get<0>(views_scratch);
	auto* src_depth_views_device = std::get<1>(views_scratch);
	auto* src_res_device = std::get<2>(views_scratch);

	CUDA_CHECK_THROW(hipMemcpyAsync(
		src_rgba_views_device,
		src_rgbd_views.data(),
		src_views.size() * sizeof(MatrixView<const uint32_t>),
		hipMemcpyHostToDevice,
		m_stream.get()
	));
	CUDA_CHECK_THROW(hipMemcpyAsync(
		src_depth_views_device, src_depth_views.data(), src_views.size() * sizeof(MatrixView<const float>), hipMemcpyHostToDevice, m_stream.get()
	));
	CUDA_CHECK_THROW(hipMemcpyAsync(src_res_device, src_res.data(), src_views.size() * sizeof(ivec2), hipMemcpyHostToDevice, m_stream.get())
	);

	auto dst_rgba_buffer = GPUImage<uint32_t>(dst_res, PM_PATCH_RADIUS, m_stream.get());
	auto dst_depth_threshold_buffer = GPUImage<float>(dst_res, PM_PATCH_RADIUS, m_stream.get());
	ivec2 padded_dst_res = dst_rgba_buffer.resolution_padded();

	default_rng_t fixed_seed_rng{0x1337};

	{
		const dim3 threads = {16, 8, 1};
		const dim3 blocks = {div_round_up((uint32_t)padded_dst_res.x, threads.x), div_round_up((uint32_t)padded_dst_res.y, threads.y), 1};

		pm_prepare_padded_dst_buffers<<<blocks, threads, 0, m_stream.get()>>>(
			padded_dst_res,
			dst_res,
			(uint32_t)src_views.size(),
			src_res_device,
			fixed_seed_rng,
			src_rgba_views_device,
			src_depth_views_device,
			dst_state_buffer.view(),
			dst_view.index_field.view(),
			dst_rgba_buffer.view(),
			dst_view.depth_buffer.view(),
			dst_depth_threshold_buffer.view(),
			dst_view.hole_mask.view()
		);

		fixed_seed_rng.advance();
	}


	reproject_viz_kernel<<<blocks, threads, 0, m_stream.get()>>>(
		dst_res,
		src_res_device,
		m_pm_enable,
		hole_labels.view(),
		dst_state_buffer.view(),
		dst_view.index_field.view(),
		dst_rgba_buffer.view(),
		dst_view.depth_buffer.view(),
		src_rgba_views_device,
		src_depth_views_device,
		MatrixView<vec4>(dst_view.render_buffer->frame_buffer(), dst_res.x, 1),
		MatrixView<float>(dst_view.render_buffer->depth_buffer(), dst_res.x, 1),
		m_pm_viz_mode,
		1.0f
	);
}

void Testbed::render(bool skip_rendering) {
	// Don't do any smoothing here if a camera path is being rendered. It'll take care
	// of the smoothing on its own.
	float frame_ms = m_camera_path.rendering ? 0.0f : m_frame_ms.val();
	apply_camera_smoothing(frame_ms);

	if (!m_render_window || !m_render || skip_rendering) {
		return;
	}

	auto start = std::chrono::steady_clock::now();
	ScopeGuard timing_guard{[&]() {
		m_render_ms.update(std::chrono::duration<float, std::milli>(std::chrono::steady_clock::now() - start).count());
	}};

	if (frobenius_norm(m_smoothed_camera - m_camera) < 0.001f) {
		m_smoothed_camera = m_camera;
	} else if (!m_camera_path.rendering) {
		reset_accumulation(true);
	}

	if (m_autofocus) {
		autofocus();
	}

	Lens lens = m_render_with_lens_distortion ? m_render_lens : Lens{};

#ifdef NGP_GUI
	if (m_hmd && m_hmd->is_visible()) {
		for (auto& view : m_views) {
			view.visualized_dimension = m_visualized_dimension;
		}

		m_n_views = {(int)m_views.size(), 1};

		m_render_with_lens_distortion = false;
		reset_accumulation(true);
	} else {
		set_n_views(1);
		m_n_views = {1, 1};

		auto& view = m_views.front();

		view.full_resolution = m_window_res;

		view.camera0 = m_smoothed_camera;

		// Motion blur over the fraction of time that the shutter is open. Interpolate in log-space to preserve rotations.
		view.camera1 = (m_camera_path.rendering && !m_gen3c_render_with_gen3c) ?
			camera_log_lerp(m_smoothed_camera, m_camera_path.render_frame_end_camera, m_camera_path.render_settings.shutter_fraction) :
			view.camera0;

		view.visualized_dimension = m_visualized_dimension;
		view.relative_focal_length = m_relative_focal_length;
		view.screen_center = m_screen_center;
		view.render_buffer->set_hidden_area_mask(nullptr);
		view.foveation = {};
		view.lens = lens;
		view.device = &primary_device();
	}

	if (m_dlss) {
		m_aperture_size = 0.0f;
		if (!m_render_lens.supports_dlss()) {
			m_render_with_lens_distortion = false;
		}
	}

	// Update dynamic res and DLSS
	{
		// Don't count the time being spent allocating buffers and resetting DLSS as part of the frame time.
		// Otherwise the dynamic resolution calculations for following frames will be thrown out of whack
		// and may even start oscillating.
		auto skip_start = std::chrono::steady_clock::now();
		ScopeGuard skip_timing_guard{[&]() { start += std::chrono::steady_clock::now() - skip_start; }};

		size_t n_pixels = 0, n_pixels_full_res = 0;
		for (const auto& view : m_views) {
			n_pixels += product(view.render_buffer->in_resolution());
			n_pixels_full_res += product(view.full_resolution);
		}

		float pixel_ratio = n_pixels == 0 ? (1.0f / 256.0f) : ((float)n_pixels / (float)n_pixels_full_res);

		float last_factor = std::sqrt(pixel_ratio);
		float factor = std::sqrt(pixel_ratio / m_render_ms.val() * 1000.0f / m_dynamic_res_target_fps);
		if (!m_dynamic_res) {
			factor = 8.f / (float)m_fixed_res_factor;
		}

		factor = clamp(factor, 1.0f / 16.0f, 1.0f);

		vec2 avg_screen_center = vec2(0.0f);
		for (size_t i = 0; i < m_views.size(); ++i) {
			avg_screen_center += m_views[i].screen_center;
		}

		avg_screen_center /= (float)m_views.size();

		for (auto&& view : m_views) {
			if (m_dlss) {
				view.render_buffer->enable_dlss(*m_dlss_provider, view.full_resolution);
			} else {
				view.render_buffer->disable_dlss();
			}

			ivec2 render_res = view.render_buffer->in_resolution();
			ivec2 new_render_res = clamp(ivec2(vec2(view.full_resolution) * factor), view.full_resolution / 16, view.full_resolution);

			if (m_camera_path.rendering && !m_gen3c_render_with_gen3c) {
				new_render_res = m_camera_path.render_settings.resolution;
			}

			float ratio = std::sqrt((float)product(render_res) / (float)product(new_render_res));
			if (ratio > 1.2f || ratio < 0.8f || factor == 1.0f || !m_dynamic_res || (m_camera_path.rendering && !m_gen3c_render_with_gen3c)) {
				render_res = new_render_res;
			}

			if (view.render_buffer->dlss()) {
				render_res = view.render_buffer->dlss()->clamp_resolution(render_res);
				view.render_buffer->dlss()->update_feature(
					render_res, view.render_buffer->dlss()->is_hdr(), view.render_buffer->dlss()->sharpen()
				);
			}

			view.render_buffer->resize(render_res);

			if (m_foveated_rendering) {
				if (m_dynamic_foveated_rendering) {
					vec2 resolution_scale = vec2(render_res) / vec2(view.full_resolution);

					// Only start foveation when DLSS if off or if DLSS is asked to do more than 1.5x upscaling.
					// The reason for the 1.5x threshold is that DLSS can do up to 3x upscaling, at which point a
					// foveation factor of 2x = 3.0x/1.5x corresponds exactly to bilinear super sampling, which is
					// helpful in suppressing DLSS's artifacts.
					float foveation_begin_factor = m_dlss ? 1.5f : 1.0f;

					resolution_scale =
						clamp(resolution_scale * foveation_begin_factor, vec2(1.0f / m_foveated_rendering_max_scaling), vec2(1.0f));
					view.foveation = {resolution_scale, vec2(1.0f) - view.screen_center, vec2(m_foveated_rendering_full_res_diameter * 0.5f)};

					m_foveated_rendering_scaling = 2.0f / sum(resolution_scale);
				} else {
					view.foveation = {
						vec2(1.0f / m_foveated_rendering_scaling),
						vec2(1.0f) - view.screen_center,
						vec2(m_foveated_rendering_full_res_diameter * 0.5f)
					};
				}
			} else {
				view.foveation = {};
			}
		}
	}

	// Make sure all in-use auxiliary GPUs have the latest model and bitfield
	std::unordered_set<CudaDevice*> devices_in_use;
	for (auto& view : m_views) {
		if (!view.device || devices_in_use.count(view.device) != 0) {
			continue;
		}

		devices_in_use.insert(view.device);
		sync_device(*view.render_buffer, *view.device);
	}

	if (m_reproject_enable) {
		render_by_reprojection(m_stream.get(), m_views);
	} else {
		SyncedMultiStream synced_streams{m_stream.get(), m_views.size()};

		std::vector<std::future<void>> futures(m_views.size());
		for (size_t i = 0; i < m_views.size(); ++i) {
			auto& view = m_views[i];
			futures[i] = view.device->enqueue_task([this, &view, stream = synced_streams.get(i)]() {
				auto device_guard = use_device(stream, *view.render_buffer, *view.device);
				render_frame_main(
					*view.device, view.camera0, view.camera1, view.screen_center, view.relative_focal_length, view.foveation, view.lens, view.visualized_dimension
				);
			});
		}

		for (size_t i = 0; i < m_views.size(); ++i) {
			auto& view = m_views[i];

			if (futures[i].valid()) {
				futures[i].get();
			}

			render_frame_epilogue(
				synced_streams.get(i),
				view.camera0,
				view.prev_camera,
				view.screen_center,
				view.relative_focal_length,
				view.foveation,
				view.prev_foveation,
				view.lens,
				*view.render_buffer,
				true
			);

			view.prev_camera = view.camera0;
			view.prev_foveation = view.foveation;
		}
	}

	for (size_t i = 0; i < m_views.size(); ++i) {
		m_rgba_render_textures.at(i)->blit_from_cuda_mapping();
		m_depth_render_textures.at(i)->blit_from_cuda_mapping();
	}

	if (m_picture_in_picture_res > 0) {
		ivec2 res{(int)m_picture_in_picture_res, (int)(m_picture_in_picture_res * 9.0f / 16.0f)};
		m_pip_render_buffer->resize(res);
		if (m_pip_render_buffer->spp() < 8) {
			// a bit gross, but let's copy the keyframe's state into the global state in order to not have to plumb
			// through the fov etc to render_frame.
			CameraKeyframe backup = copy_camera_to_keyframe();
			CameraKeyframe pip_kf = m_camera_path.eval_camera_path(m_camera_path.play_time);
			set_camera_from_keyframe(pip_kf);

			if (m_reproject_enable) {
				std::vector<View> views(1);
				auto& view = views.front();
				view.camera0 = pip_kf.m();
				view.camera1 = pip_kf.m();
				view.prev_camera = pip_kf.m();
				view.screen_center = m_screen_center;
				view.relative_focal_length = m_relative_focal_length;
				view.foveation = {};
				view.prev_foveation = {};
				view.lens = lens;
				view.visualized_dimension = m_visualized_dimension;
				view.render_buffer = m_pip_render_buffer;

				render_by_reprojection(m_stream.get(), views);
			} else {
				render_frame(
					m_stream.get(),
					pip_kf.m(),
					pip_kf.m(),
					pip_kf.m(),
					m_screen_center,
					m_relative_focal_length,
					{}, // foveation
					{}, // prev foveation
					lens,
					m_visualized_dimension,
					*m_pip_render_buffer
				);
			}

			set_camera_from_keyframe(backup);
			m_pip_render_texture->blit_from_cuda_mapping();
		}
	}
#endif

	CUDA_CHECK_THROW(hipStreamSynchronize(m_stream.get()));
}

mat4x3 Testbed::view_camera(size_t view_idx) const {
	if (m_views.size() <= view_idx) {
		throw std::runtime_error{fmt::format("View #{} does not exist.", view_idx)};
	}

	auto& view = m_views.at(view_idx);
	return view.camera0;
}


#ifdef NGP_GUI
void Testbed::create_second_window() {
	if (m_second_window.window) {
		return;
	}

	bool frameless = false;
	glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
	glfwWindowHint(GLFW_RESIZABLE, !frameless);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
	glfwWindowHint(GLFW_CENTER_CURSOR, false);
	glfwWindowHint(GLFW_DECORATED, !frameless);
	glfwWindowHint(GLFW_SCALE_TO_MONITOR, frameless);
	glfwWindowHint(GLFW_TRANSPARENT_FRAMEBUFFER, true);
	// get the window size / coordinates
	int win_w = 0, win_h = 0, win_x = 0, win_y = 0;
	GLuint ps = 0, vs = 0;

	{
		win_w = 1920;
		win_h = 1080;
		win_x = 0x40000000;
		win_y = 0x40000000;
		static const char* copy_shader_vert =
			"\
			in vec2 vertPos_data;\n\
			out vec2 texCoords;\n\
			void main(){\n\
				gl_Position = vec4(vertPos_data.xy, 0.0, 1.0);\n\
				texCoords = (vertPos_data.xy + 1.0) * 0.5; texCoords.y=1.0-texCoords.y;\n\
			}";
		static const char* copy_shader_frag =
			"\
			in vec2 texCoords;\n\
			out vec4 fragColor;\n\
			uniform sampler2D screenTex;\n\
			void main(){\n\
				fragColor = texture(screenTex, texCoords.xy);\n\
			}";
		vs = compile_shader(false, copy_shader_vert);
		ps = compile_shader(true, copy_shader_frag);
	}

	m_second_window.window = glfwCreateWindow(win_w, win_h, "Fullscreen Output", NULL, m_glfw_window);
	if (win_x != 0x40000000) {
		glfwSetWindowPos(m_second_window.window, win_x, win_y);
	}

	glfwMakeContextCurrent(m_second_window.window);
	m_second_window.program = glCreateProgram();
	glAttachShader(m_second_window.program, vs);
	glAttachShader(m_second_window.program, ps);
	glLinkProgram(m_second_window.program);
	if (!check_shader(m_second_window.program, "shader program", true)) {
		glDeleteProgram(m_second_window.program);
		m_second_window.program = 0;
	}

	// vbo and vao
	glGenVertexArrays(1, &m_second_window.vao);
	glGenBuffers(1, &m_second_window.vbo);
	glBindVertexArray(m_second_window.vao);
	const float fsquadVerts[] = {-1.0f, -1.0f, -1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, -1.0f, -1.0f, -1.0f};
	glBindBuffer(GL_ARRAY_BUFFER, m_second_window.vbo);
	glBufferData(GL_ARRAY_BUFFER, sizeof(fsquadVerts), fsquadVerts, GL_STATIC_DRAW);
	glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 2 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(0);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	glBindVertexArray(0);
}

void Testbed::set_n_views(size_t n_views) {
	bool changed_views = n_views != m_views.size();

	while (m_views.size() > n_views) {
		m_views.pop_back();
	}

	m_rgba_render_textures.resize(n_views);
	m_depth_render_textures.resize(n_views);

	while (m_views.size() < n_views) {
		size_t idx = m_views.size();
		m_rgba_render_textures[idx] = std::make_shared<GLTexture>();
		m_depth_render_textures[idx] = std::make_shared<GLTexture>();
		m_views.emplace_back(View{std::make_shared<CudaRenderBuffer>(m_rgba_render_textures[idx], m_depth_render_textures[idx])});
	}

};
#endif // NGP_GUI

void Testbed::init_window(int resw, int resh, bool hidden, bool second_window) {
#ifndef NGP_GUI
	throw std::runtime_error{"init_window failed: NGP was built without GUI support"};
#else
	m_window_res = {resw, resh};

	glfwSetErrorCallback(glfw_error_callback);
	if (!glfwInit()) {
		throw std::runtime_error{"GLFW could not be initialized."};
	}

#	ifdef NGP_VULKAN
	// Only try to initialize DLSS (Vulkan+NGX) if the
	// GPU is sufficiently new. Older GPUs don't support
	// DLSS, so it is preferable to not make a futile
	// attempt and emit a warning that confuses users.
	if (primary_device().compute_capability() >= 70) {
		try {
			m_dlss_provider = init_vulkan_and_ngx();
		} catch (const std::runtime_error& e) {
			tlog::warning() << "Could not initialize Vulkan and NGX. DLSS not supported. (" << e.what() << ")";
		}
	}
#	endif

	glfwWindowHint(GLFW_VISIBLE, hidden ? GLFW_FALSE : GLFW_TRUE);
	std::string title = "Gen3C GUI";
	m_glfw_window = glfwCreateWindow(m_window_res.x, m_window_res.y, title.c_str(), NULL, NULL);
	if (m_glfw_window == NULL) {
		throw std::runtime_error{"GLFW window could not be created."};
	}
	glfwMakeContextCurrent(m_glfw_window);
#	ifdef _WIN32
	if (gl3wInit()) {
		throw std::runtime_error{"GL3W could not be initialized."};
	}
#	else
	glewExperimental = 1;
	if (glewInit()) {
		throw std::runtime_error{"GLEW could not be initialized."};
	}
#	endif
	glfwSwapInterval(m_vsync ? 1 : 0); // Disable vsync

	GLint gl_version_minor, gl_version_major;
	glGetIntegerv(GL_MINOR_VERSION, &gl_version_minor);
	glGetIntegerv(GL_MAJOR_VERSION, &gl_version_major);

	if (gl_version_major < 3 || (gl_version_major == 3 && gl_version_minor < 1)) {
		throw std::runtime_error{
			fmt::format("Unsupported OpenGL version {}.{}. Gen3C requires at least OpenGL 3.1", gl_version_major, gl_version_minor)
		};
	}

	tlog::success() << "Initialized OpenGL version " << glGetString(GL_VERSION);

	glfwSetWindowUserPointer(m_glfw_window, this);
	glfwSetDropCallback(m_glfw_window, [](GLFWwindow* window, int count, const char** paths) {
		Testbed* testbed = (Testbed*)glfwGetWindowUserPointer(window);
		if (!testbed) {
			return;
		}

		if (testbed->m_file_drop_callback) {
			if (testbed->m_file_drop_callback(std::vector<std::string>(paths, paths + count))) {
				// Files were handled by the callback.
				return;
			}
		}

		for (int i = 0; i < count; i++) {
			testbed->load_file(paths[i]);
		}
	});

	glfwSetKeyCallback(m_glfw_window, [](GLFWwindow* window, int key, int scancode, int action, int mods) {
		Testbed* testbed = (Testbed*)glfwGetWindowUserPointer(window);
		if (testbed) {
			testbed->redraw_gui_next_frame();
		}
	});

	glfwSetCursorPosCallback(m_glfw_window, [](GLFWwindow* window, double xpos, double ypos) {
		Testbed* testbed = (Testbed*)glfwGetWindowUserPointer(window);
		if (testbed && (ImGui::IsAnyItemActive() || ImGui::GetIO().WantCaptureMouse || ImGuizmo::IsUsing()) &&
			(ImGui::GetIO().MouseDown[0] || ImGui::GetIO().MouseDown[1] || ImGui::GetIO().MouseDown[2])) {
			testbed->redraw_gui_next_frame();
		}
	});

	glfwSetMouseButtonCallback(m_glfw_window, [](GLFWwindow* window, int button, int action, int mods) {
		Testbed* testbed = (Testbed*)glfwGetWindowUserPointer(window);
		if (testbed) {
			testbed->redraw_gui_next_frame();
		}
	});

	glfwSetScrollCallback(m_glfw_window, [](GLFWwindow* window, double xoffset, double yoffset) {
		Testbed* testbed = (Testbed*)glfwGetWindowUserPointer(window);
		if (testbed) {
			testbed->redraw_gui_next_frame();
		}
	});

	glfwSetWindowSizeCallback(m_glfw_window, [](GLFWwindow* window, int width, int height) {
		Testbed* testbed = (Testbed*)glfwGetWindowUserPointer(window);
		if (testbed) {
			testbed->redraw_next_frame();
		}
	});

	glfwSetFramebufferSizeCallback(m_glfw_window, [](GLFWwindow* window, int width, int height) {
		Testbed* testbed = (Testbed*)glfwGetWindowUserPointer(window);
		if (testbed) {
			testbed->redraw_next_frame();
		}
	});

	float xscale, yscale;
	glfwGetWindowContentScale(m_glfw_window, &xscale, &yscale);

	// IMGUI init
	IMGUI_CHECKVERSION();
	ImGui::CreateContext();
	ImGuiIO& io = ImGui::GetIO();
	(void)io;

	// By default, imgui places its configuration (state of the GUI -- size of windows, which regions are expanded, etc.) in ./imgui.ini
	// relative to the working directory. Instead, we would like to place imgui.ini in the directory that Gen3C project resides in.
	static std::string ini_filename;
	ini_filename = (root_dir() / "imgui.ini").str();
	io.IniFilename = ini_filename.c_str();

	// New ImGui event handling seems to make camera controls laggy if input trickling is true. So disable input trickling.
	io.ConfigInputTrickleEventQueue = false;
	ImGui::StyleColorsDark();
	ImGui_ImplGlfw_InitForOpenGL(m_glfw_window, true);
	ImGui_ImplOpenGL3_Init("#version 140");

	ImGui::GetStyle().ScaleAllSizes(xscale);
	ImFontConfig font_cfg;
	font_cfg.SizePixels = 13.0f * xscale;
	io.Fonts->AddFontDefault(&font_cfg);
	ImFontConfig overlay_font_cfg;
	overlay_font_cfg.SizePixels = 128.0f * xscale;
	m_imgui.overlay_font = io.Fonts->AddFontDefault(&overlay_font_cfg);

	init_opengl_shaders();

	// Make sure there's at least one usable render texture
	set_n_views(1);
	m_views.front().full_resolution = m_window_res;
	m_views.front().render_buffer->resize(m_views.front().full_resolution);

	m_pip_render_texture = std::make_shared<GLTexture>();
	m_pip_render_buffer = std::make_shared<CudaRenderBuffer>(m_pip_render_texture);

	m_render_window = true;

	if (m_second_window.window == nullptr && second_window) {
		create_second_window();
	}
#endif // NGP_GUI
}

void Testbed::destroy_window() {
#ifndef NGP_GUI
	throw std::runtime_error{"destroy_window failed: NGP was built without GUI support"};
#else
	if (!m_render_window) {
		throw std::runtime_error{"Window must be initialized to be destroyed."};
	}

	m_hmd.reset();

	m_views.clear();
	m_rgba_render_textures.clear();
	m_depth_render_textures.clear();

	m_pip_render_buffer.reset();
	m_pip_render_texture.reset();

	m_dlss = false;
	m_dlss_provider.reset();

	ImGui_ImplOpenGL3_Shutdown();
	ImGui_ImplGlfw_Shutdown();
	ImGui::DestroyContext();
	glfwDestroyWindow(m_glfw_window);
	glfwTerminate();

	m_blit_program = 0;
	m_blit_vao = 0;

	m_glfw_window = nullptr;
	m_render_window = false;
#endif // NGP_GUI
}

void Testbed::init_vr() {
#ifndef NGP_GUI
	throw std::runtime_error{"init_vr failed: NGP was built without GUI support"};
#else
	try {
		if (!m_glfw_window) {
			throw std::runtime_error{"`init_window` must be called before `init_vr`"};
		}

#	if defined(XR_USE_PLATFORM_WIN32)
		m_hmd = std::make_unique<OpenXRHMD>(wglGetCurrentDC(), glfwGetWGLContext(m_glfw_window));
#	elif defined(XR_USE_PLATFORM_XLIB)
		Display* xDisplay = glfwGetX11Display();
		GLXContext glxContext = glfwGetGLXContext(m_glfw_window);

		int glxFBConfigXID = 0;
		glXQueryContext(xDisplay, glxContext, GLX_FBCONFIG_ID, &glxFBConfigXID);
		int attributes[3] = {GLX_FBCONFIG_ID, glxFBConfigXID, 0};
		int nelements = 1;
		GLXFBConfig* pglxFBConfig = glXChooseFBConfig(xDisplay, 0, attributes, &nelements);
		if (nelements != 1 || !pglxFBConfig) {
			throw std::runtime_error{"init_vr(): Couldn't obtain GLXFBConfig"};
		}

		GLXFBConfig glxFBConfig = *pglxFBConfig;

		XVisualInfo* visualInfo = glXGetVisualFromFBConfig(xDisplay, glxFBConfig);
		if (!visualInfo) {
			throw std::runtime_error{"init_vr(): Couldn't obtain XVisualInfo"};
		}

		m_hmd = std::make_unique<OpenXRHMD>(xDisplay, visualInfo->visualid, glxFBConfig, glXGetCurrentDrawable(), glxContext);
#	elif defined(XR_USE_PLATFORM_WAYLAND)
		m_hmd = std::make_unique<OpenXRHMD>(glfwGetWaylandDisplay());
#	endif

		// Enable aggressive optimizations to make the VR experience smooth.
		update_vr_performance_settings();

		// If multiple GPUs are available, shoot for 60 fps in VR.
		// Otherwise, it wouldn't be realistic to expect more than 30.
		m_dynamic_res_target_fps = m_devices.size() > 1 ? 60 : 30;
		m_background_color = {0.0f, 0.0f, 0.0f, 0.0f};
	} catch (const std::runtime_error& e) {
		if (std::string{e.what()}.find("XR_ERROR_FORM_FACTOR_UNAVAILABLE") != std::string::npos) {
			throw std::runtime_error{
				"Could not initialize VR. Ensure that SteamVR, OculusVR, or any other OpenXR-compatible runtime is running. Also set it as the active OpenXR runtime."
			};
		} else {
			throw std::runtime_error{fmt::format("Could not initialize VR: {}", e.what())};
		}
	}
#endif // NGP_GUI
}

void Testbed::update_vr_performance_settings() {
#ifdef NGP_GUI
	if (m_hmd) {
		auto blend_mode = m_hmd->environment_blend_mode();

		// DLSS is instrumental in getting VR to look good. Enable if possible.
		// If the environment is blended in (such as in XR/AR applications),
		// DLSS causes jittering at object sillhouettes (doesn't deal well with alpha),
		// and hence stays disabled.
		m_dlss = (blend_mode == EEnvironmentBlendMode::Opaque) && m_dlss_provider;

		// Foveated rendering is similarly vital in getting high performance without losing
		// resolution in the middle of the view.
		m_foveated_rendering = true;

		// Many VR runtimes perform optical flow for automatic reprojection / motion smoothing.
		// This breaks down for solid-color background, sometimes leading to artifacts. Hence:
		// set background color to transparent and, in spherical_checkerboard_kernel(...),
		// blend a checkerboard. If the user desires a solid background nonetheless, they can
		// set the background color to have an alpha value of 1.0 manually via the GUI or via Python.
		m_render_transparency_as_checkerboard = (blend_mode == EEnvironmentBlendMode::Opaque);
	} else {
		m_foveated_rendering = false;
		m_render_transparency_as_checkerboard = false;
	}
#endif // NGP_GUI
}

bool Testbed::frame() {
#ifdef NGP_GUI
	if (m_render_window) {
		if (!begin_frame()) {
			return false;
		}

		handle_user_input();
		begin_vr_frame_and_handle_vr_input();
	}
#endif

	bool skip_rendering = false;
	if (!m_dlss && m_max_spp > 0 && !m_views.empty() && m_views.front().render_buffer->spp() >= m_max_spp) {
		skip_rendering = true;
	}

	if (m_camera_path.rendering && !m_gen3c_render_with_gen3c) {
		prepare_next_camera_path_frame();
		skip_rendering = false;
	}

	if (m_record_camera_path && !m_views.empty()) {
		m_camera_path.spline_order = 1;
		const float timestamp = m_camera_path.duration_seconds() + m_frame_ms.val() / 1000.0f;
		m_camera_path.add_camera(m_views[0].camera0, focal_length_to_fov(1.0f, m_views[0].relative_focal_length[m_fov_axis]), timestamp);

		m_camera_path.keyframe_subsampling = (int)m_camera_path.keyframes.size();
		m_camera_path.editing_kernel_type = EEditingKernel::Gaussian;
	}

#ifdef NGP_GUI
	if (m_hmd && m_hmd->is_visible()) {
		skip_rendering = false;
	}
#endif

	if (!skip_rendering || std::chrono::steady_clock::now() - m_last_gui_draw_time_point > 50ms) {
		redraw_gui_next_frame();
	}

	try {
		while (true) {
			(*m_task_queue.tryPop())();
		}
	} catch (const SharedQueueEmptyException&) {}

	render(skip_rendering);

#ifdef NGP_GUI
	if (m_render_window) {
		if (m_gui_redraw) {
			draw_gui();
			m_gui_redraw = false;

			m_last_gui_draw_time_point = std::chrono::steady_clock::now();
		}

		ImGui::EndFrame();
	}

	if (m_hmd && m_vr_frame_info) {
		// If HMD is visible to the user, splat rendered images to the HMD
		if (m_hmd->is_visible()) {
			size_t n_views = std::min(m_views.size(), m_vr_frame_info->views.size());

			// Blit textures to the OpenXR-owned framebuffers (each corresponding to one eye)
			for (size_t i = 0; i < n_views; ++i) {
				const auto& vr_view = m_vr_frame_info->views.at(i);

				ivec2 resolution = {
					vr_view.view.subImage.imageRect.extent.width,
					vr_view.view.subImage.imageRect.extent.height,
				};

				blit_texture(
					m_views.at(i).foveation,
					m_rgba_render_textures.at(i)->texture(),
					GL_LINEAR,
					m_depth_render_textures.at(i)->texture(),
					vr_view.framebuffer,
					ivec2(0),
					resolution
				);
			}

			glFinish();
		}

		// Far and near planes are intentionally reversed, because we map depth inversely
		// to z. I.e. a window-space depth of 1 refers to the near plane and a depth of 0
		// to the far plane. This results in much better numeric precision.
		m_hmd->end_frame(m_vr_frame_info, m_ndc_zfar / m_scale, m_ndc_znear / m_scale, m_vr_use_depth_reproject);
	}
#endif

	return true;
}

bool Testbed::want_repl() {
	bool b = m_want_repl;
	m_want_repl = false;
	return b;
}

void Testbed::apply_camera_smoothing(float elapsed_ms) {
	// Ensure our camera rotation remains an orthogonal matrix as numeric
	// errors accumulate across frames.
	m_camera = orthogonalize(m_camera);

	if (m_camera_smoothing) {
		float decay = std::pow(0.02f, elapsed_ms / 1000.0f);
		m_smoothed_camera = orthogonalize(camera_log_lerp(m_smoothed_camera, m_camera, 1.0f - decay));
	} else {
		m_smoothed_camera = m_camera;
	}
}

CameraKeyframe Testbed::copy_camera_to_keyframe() const { return CameraKeyframe(m_camera, fov(), 0.0f); }

void Testbed::set_camera_from_keyframe(const CameraKeyframe& k) {
	m_camera = k.m();
	set_fov(k.fov);
}

void Testbed::set_camera_from_time(float t) {
	if (m_camera_path.keyframes.empty()) {
		return;
	}

	set_camera_from_keyframe(m_camera_path.eval_camera_path(t));
}

float Testbed::fov() const { return focal_length_to_fov(1.0f, m_relative_focal_length[m_fov_axis]); }

void Testbed::set_fov(float val) { m_relative_focal_length = vec2(fov_to_focal_length(1, val)); }

vec2 Testbed::fov_xy() const { return focal_length_to_fov(ivec2(1), m_relative_focal_length); }

void Testbed::set_fov_xy(const vec2& val) { m_relative_focal_length = fov_to_focal_length(ivec2(1), val); }

Testbed::Testbed(ETestbedMode mode) {
	tcnn::set_log_callback([](LogSeverity severity, const std::string& msg) {
		tlog::ESeverity s = tlog::ESeverity::Info;
		switch (severity) {
			case LogSeverity::Info: s = tlog::ESeverity::Info; break;
			case LogSeverity::Debug: s = tlog::ESeverity::Debug; break;
			case LogSeverity::Warning: s = tlog::ESeverity::Warning; break;
			case LogSeverity::Error: s = tlog::ESeverity::Error; break;
			case LogSeverity::Success: s = tlog::ESeverity::Success; break;
			default: break;
		}
		tlog::log(s) << msg;
	});

	if (!(__CUDACC_VER_MAJOR__ > 10 || (__CUDACC_VER_MAJOR__ == 10 && __CUDACC_VER_MINOR__ >= 2))) {
		throw std::runtime_error{"Testbed requires CUDA 10.2 or later."};
	}

#ifdef NGP_GUI
	// Ensure we're running on the GPU that'll host our GUI. To do so, try creating a dummy
	// OpenGL context, figure out the GPU it's running on, and then kill that context again.
	if (!is_wsl() && glfwInit()) {
		glfwWindowHint(GLFW_VISIBLE, GLFW_FALSE);
		GLFWwindow* offscreen_context = glfwCreateWindow(640, 480, "", NULL, NULL);

		if (offscreen_context) {
			glfwMakeContextCurrent(offscreen_context);

			int gl_device = -1;
			unsigned int device_count = 0;
			if (hipGLGetDevices(&device_count, &gl_device, 1, hipGLDeviceListAll) == hipSuccess) {
				if (device_count > 0 && gl_device >= 0) {
					set_cuda_device(gl_device);
				}
			}

			glfwDestroyWindow(offscreen_context);
		}

		glfwTerminate();
	}
#endif

	// Reset our stream, which was allocated on the originally active device,
	// to make sure it corresponds to the now active device.
	m_stream = {};

	int active_device = cuda_device();
	int active_compute_capability = cuda_compute_capability();
	tlog::success() << fmt::format(
		"Initialized CUDA {}. Active GPU is #{}: {} [{}]", cuda_runtime_version_string(), active_device, cuda_device_name(), active_compute_capability
	);

	if (active_compute_capability < MIN_GPU_ARCH) {
		tlog::warning() << "Insufficient compute capability " << active_compute_capability << " detected.";
		tlog::warning() << "This program was compiled for >=" << MIN_GPU_ARCH << " and may thus behave unexpectedly.";
	}

	m_devices.emplace_back(active_device, true);

	int n_devices = cuda_device_count();
	for (int i = 0; i < n_devices; ++i) {
		if (i == active_device) {
			continue;
		}

		if (cuda_compute_capability(i) >= MIN_GPU_ARCH) {
			m_devices.emplace_back(i, false);
		}
	}

	if (m_devices.size() > 1) {
		tlog::success() << "Detected auxiliary GPUs:";
		for (size_t i = 1; i < m_devices.size(); ++i) {
			const auto& device = m_devices[i];
			tlog::success() << "  #" << device.id() << ": " << device.name() << " [" << device.compute_capability() << "]";
		}
	}

	set_mode(mode);
	set_exposure(0);

	reset_camera();
}

Testbed::~Testbed() {
	// If any temporary file was created, make sure it's deleted
	clear_tmp_dir();

	if (m_render_window) {
		destroy_window();
	}
}

bool Testbed::clear_tmp_dir() {
	wait_all(m_render_futures);
	m_render_futures.clear();

	bool success = true;
	auto tmp_dir = fs::path{"tmp"};
	if (tmp_dir.exists()) {
		if (tmp_dir.is_directory()) {
			for (const auto& path : fs::directory{tmp_dir}) {
				if (path.is_file()) {
					success &= path.remove_file();
				}
			}
		}

		success &= tmp_dir.remove_file();
	}

	return success;
}

vec2 Testbed::calc_focal_length(const ivec2& resolution, const vec2& relative_focal_length, int fov_axis, float zoom) const {
	return relative_focal_length * (float)resolution[fov_axis] * zoom;
}

vec2 Testbed::render_screen_center(const vec2& screen_center) const {
	// see pixel_to_ray for how screen center is used; 0.5, 0.5 is 'normal'. we flip so that it becomes the point in the
	// original image we want to center on.
	return (0.5f - screen_center) * m_zoom + 0.5f;
}

__global__ void dlss_prep_kernel(
	ivec2 resolution,
	uint32_t sample_index,
	vec2 focal_length,
	vec2 screen_center,
	vec3 parallax_shift,
	bool snap_to_pixel_centers,
	float* depth_buffer,
	const float znear,
	const float zfar,
	mat4x3 camera,
	mat4x3 prev_camera,
	hipSurfaceObject_t depth_surface,
	hipSurfaceObject_t mvec_surface,
	hipSurfaceObject_t exposure_surface,
	Foveation foveation,
	Foveation prev_foveation,
	Lens lens
) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= resolution.x || y >= resolution.y) {
		return;
	}

	uint32_t idx = x + resolution.x * y;

	uint32_t x_orig = x;
	uint32_t y_orig = y;

	const float depth = depth_buffer[idx];
	vec2 mvec = motion_vector(
		sample_index,
		{(int)x, (int)y},
		resolution,
		focal_length,
		camera,
		prev_camera,
		screen_center,
		parallax_shift,
		snap_to_pixel_centers,
		depth,
		foveation,
		prev_foveation,
		lens
	);

	surf2Dwrite(make_float2(mvec.x, mvec.y), mvec_surface, x_orig * sizeof(float2), y_orig);

	// DLSS was trained on games, which presumably used standard normalized device coordinates (ndc)
	// depth buffers. So: convert depth to NDC with reasonable near- and far planes.
	surf2Dwrite(to_ndc_depth(depth, znear, zfar), depth_surface, x_orig * sizeof(float), y_orig);

	// First thread write an exposure factor of 1. Since DLSS will run on tonemapped data,
	// exposure is assumed to already have been applied to DLSS' inputs.
	if (x_orig == 0 && y_orig == 0) {
		surf2Dwrite(1.0f, exposure_surface, 0, 0);
	}
}

__global__ void spherical_checkerboard_kernel(
	ivec2 resolution,
	vec2 focal_length,
	mat4x3 camera,
	vec2 screen_center,
	vec3 parallax_shift,
	Foveation foveation,
	Lens lens,
	vec4 background_color,
	vec4* frame_buffer
) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= resolution.x || y >= resolution.y) {
		return;
	}

	Ray ray = pixel_to_ray(
		0,
		{(int)x, (int)y},
		resolution,
		focal_length,
		camera,
		screen_center,
		parallax_shift,
		false,
		0.0f,
		1.0f,
		0.0f,
		foveation,
		{}, // No need for hidden area mask
		lens
	);

	// Blend with checkerboard to break up reprojection weirdness in some VR runtimes
	host_device_swap(ray.d.z, ray.d.y);
	vec2 spherical = dir_to_spherical(normalize(ray.d)) * 32.0f / PI();
	const vec4 dark_gray = {0.5f, 0.5f, 0.5f, 1.0f};
	const vec4 light_gray = {0.55f, 0.55f, 0.55f, 1.0f};
	vec4 checker = fabsf(fmodf(floorf(spherical.x) + floorf(spherical.y), 2.0f)) < 0.5f ? dark_gray : light_gray;

	// Blend background color on top of checkerboard first (checkerboard is meant to be "behind" the background,
	// representing transparency), and then blend the result behind the frame buffer.
	background_color.rgb() = srgb_to_linear(background_color.rgb());
	background_color += (1.0f - background_color.a) * checker;

	uint32_t idx = x + resolution.x * y;
	frame_buffer[idx] += (1.0f - frame_buffer[idx].a) * background_color;
}

__global__ void vr_overlay_hands_kernel(
	ivec2 resolution,
	vec2 focal_length,
	mat4x3 camera,
	vec2 screen_center,
	vec3 parallax_shift,
	Foveation foveation,
	Lens lens,
	vec3 left_hand_pos,
	float left_grab_strength,
	vec4 left_hand_color,
	vec3 right_hand_pos,
	float right_grab_strength,
	vec4 right_hand_color,
	float hand_radius,
	EColorSpace output_color_space,
	hipSurfaceObject_t surface
	// TODO: overwrite depth buffer
) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= resolution.x || y >= resolution.y) {
		return;
	}

	Ray ray = pixel_to_ray(
		0,
		{(int)x, (int)y},
		resolution,
		focal_length,
		camera,
		screen_center,
		parallax_shift,
		false,
		0.0f,
		1.0f,
		0.0f,
		foveation,
		{}, // No need for hidden area mask
		lens
	);

	vec4 color = vec4(0.0f);
	auto composit_hand = [&](vec3 hand_pos, float grab_strength, vec4 hand_color) {
		// Don't render the hand indicator if it's behind the ray origin.
		if (dot(ray.d, hand_pos - ray.o) < 0.0f) {
			return;
		}

		float distance = ray.distance_to(hand_pos);

		vec4 base_color = vec4(0.0f);
		const vec4 border_color = {0.4f, 0.4f, 0.4f, 0.4f};

		// Divide hand radius into an inner part (4/5ths) and a border (1/5th).
		float radius = hand_radius * 0.8f;
		float border_width = hand_radius * 0.2f;

		// When grabbing, shrink the inner part as a visual indicator.
		radius *= 0.5f + 0.5f * (1.0f - grab_strength);

		if (distance < radius) {
			base_color = hand_color;
		} else if (distance < radius + border_width) {
			base_color = border_color;
		} else {
			return;
		}

		// Make hand color opaque when grabbing.
		base_color.a = grab_strength + (1.0f - grab_strength) * base_color.a;
		color += base_color * (1.0f - color.a);
	};

	if (dot(ray.d, left_hand_pos - ray.o) < dot(ray.d, right_hand_pos - ray.o)) {
		composit_hand(left_hand_pos, left_grab_strength, left_hand_color);
		composit_hand(right_hand_pos, right_grab_strength, right_hand_color);
	} else {
		composit_hand(right_hand_pos, right_grab_strength, right_hand_color);
		composit_hand(left_hand_pos, left_grab_strength, left_hand_color);
	}

	// Blend with existing color of pixel
	vec4 prev_color;
	surf2Dread((float4*)&prev_color, surface, x * sizeof(float4), y);
	if (output_color_space == EColorSpace::SRGB) {
		prev_color.rgb() = srgb_to_linear(prev_color.rgb());
	}

	color += (1.0f - color.a) * prev_color;

	if (output_color_space == EColorSpace::SRGB) {
		color.rgb() = linear_to_srgb(color.rgb());
	}

	surf2Dwrite(to_float4(color), surface, x * sizeof(float4), y);
}

void Testbed::render_by_reprojection(hipStream_t stream, std::vector<View>& views) {
	// Reprojection from view cache
	int n_src_views = std::max(std::min(m_reproject_max_src_view_index, (int)m_reproject_src_views.size()) - m_reproject_min_src_view_index, 0);

	std::vector<const View*> src_views(n_src_views);
	for (int i = 0; i < n_src_views; ++i) {
		// Invert order of src views to reproject from the most recent one first and fill in the holes / closer content with older views.
		src_views[n_src_views - i - 1] = &m_reproject_src_views[i + m_reproject_min_src_view_index];
	}

	for (size_t i = 0; i < views.size(); ++i) {
		auto& view = views[i];

		reproject_views(src_views, view);

		render_frame_epilogue(
			stream,
			view.camera0,
			view.prev_camera,
			view.screen_center,
			view.relative_focal_length,
			view.foveation,
			view.prev_foveation,
			view.lens,
			*view.render_buffer,
			true
		);

		view.prev_camera = view.camera0;
		view.prev_foveation = view.foveation;
	}
}

void Testbed::render_frame(
	hipStream_t stream,
	const mat4x3& camera_matrix0,
	const mat4x3& camera_matrix1,
	const mat4x3& prev_camera_matrix,
	const vec2& orig_screen_center,
	const vec2& relative_focal_length,
	const Foveation& foveation,
	const Foveation& prev_foveation,
	const Lens& lens,
	int visualized_dimension,
	CudaRenderBuffer& render_buffer,
	bool to_srgb,
	CudaDevice* device
) {
	if (!device) {
		device = &primary_device();
	}

	sync_device(render_buffer, *device);

	{
		auto device_guard = use_device(stream, render_buffer, *device);
		render_frame_main(
			*device, camera_matrix0, camera_matrix1, orig_screen_center, relative_focal_length, foveation, lens, visualized_dimension
		);
	}

	render_frame_epilogue(
		stream, camera_matrix0, prev_camera_matrix, orig_screen_center, relative_focal_length, foveation, prev_foveation, lens, render_buffer, to_srgb
	);
}

void Testbed::render_frame_main(
	CudaDevice& device,
	const mat4x3& camera_matrix0,
	const mat4x3& camera_matrix1,
	const vec2& orig_screen_center,
	const vec2& relative_focal_length,
	const Foveation& foveation,
	const Lens& lens,
	int visualized_dimension
) {
	device.render_buffer_view().clear(device.stream());

	vec2 focal_length = calc_focal_length(device.render_buffer_view().resolution, relative_focal_length, m_fov_axis, m_zoom);
	vec2 screen_center = render_screen_center(orig_screen_center);
}

void Testbed::render_frame_epilogue(
	hipStream_t stream,
	const mat4x3& camera_matrix0,
	const mat4x3& prev_camera_matrix,
	const vec2& orig_screen_center,
	const vec2& relative_focal_length,
	const Foveation& foveation,
	const Foveation& prev_foveation,
	const Lens& lens,
	CudaRenderBuffer& render_buffer,
	bool to_srgb
) {
	vec2 focal_length = calc_focal_length(render_buffer.in_resolution(), relative_focal_length, m_fov_axis, m_zoom);
	vec2 screen_center = render_screen_center(orig_screen_center);

	render_buffer.set_color_space(m_color_space);
	render_buffer.set_tonemap_curve(m_tonemap_curve);

	// Prepare DLSS data: motion vectors, scaled depth, exposure
	if (render_buffer.dlss()) {
		auto res = render_buffer.in_resolution();

		const dim3 threads = {16, 8, 1};
		const dim3 blocks = {div_round_up((uint32_t)res.x, threads.x), div_round_up((uint32_t)res.y, threads.y), 1};

		dlss_prep_kernel<<<blocks, threads, 0, stream>>>(
			res,
			render_buffer.spp(),
			focal_length,
			screen_center,
			m_parallax_shift,
			m_snap_to_pixel_centers,
			render_buffer.depth_buffer(),
			m_ndc_znear,
			m_ndc_zfar,
			camera_matrix0,
			prev_camera_matrix,
			render_buffer.dlss()->depth(),
			render_buffer.dlss()->mvec(),
			render_buffer.dlss()->exposure(),
			foveation,
			prev_foveation,
			lens
		);

		render_buffer.set_dlss_sharpening(m_dlss_sharpening);
	}

	EColorSpace output_color_space = to_srgb ? EColorSpace::SRGB : EColorSpace::Linear;

	if (m_render_transparency_as_checkerboard) {
		mat4x3 checkerboard_transform = mat4x3::identity();

#ifdef NGP_GUI
		if (m_hmd && m_vr_frame_info && !m_vr_frame_info->views.empty()) {
			checkerboard_transform = m_vr_frame_info->views[0].pose;
		}
#endif

		auto res = render_buffer.in_resolution();
		const dim3 threads = {16, 8, 1};
		const dim3 blocks = {div_round_up((uint32_t)res.x, threads.x), div_round_up((uint32_t)res.y, threads.y), 1};
		spherical_checkerboard_kernel<<<blocks, threads, 0, stream>>>(
			res,
			focal_length,
			checkerboard_transform,
			screen_center,
			m_parallax_shift,
			foveation,
			lens,
			m_background_color,
			render_buffer.frame_buffer()
		);
	}

	render_buffer.accumulate(m_exposure, stream);
	render_buffer.tonemap(m_exposure, m_background_color, output_color_space, m_ndc_znear, m_ndc_zfar, m_snap_to_pixel_centers, stream);

#ifdef NGP_GUI
	// If in VR, indicate the hand position and render transparent background
	if (m_hmd && m_vr_frame_info) {
		auto& hands = m_vr_frame_info->hands;

		auto res = render_buffer.out_resolution();
		const dim3 threads = {16, 8, 1};
		const dim3 blocks = {div_round_up((uint32_t)res.x, threads.x), div_round_up((uint32_t)res.y, threads.y), 1};
		vr_overlay_hands_kernel<<<blocks, threads, 0, stream>>>(
			res,
			focal_length * vec2(render_buffer.out_resolution()) / vec2(render_buffer.in_resolution()),
			camera_matrix0,
			screen_center,
			m_parallax_shift,
			foveation,
			lens,
			vr_to_world(hands[0].pose[3]),
			hands[0].grab_strength,
			{hands[0].pressing ? 0.8f : 0.0f, 0.0f, 0.0f, 0.8f},
			vr_to_world(hands[1].pose[3]),
			hands[1].grab_strength,
			{hands[1].pressing ? 0.8f : 0.0f, 0.0f, 0.0f, 0.8f},
			0.05f * m_scale, // Hand radius
			output_color_space,
			render_buffer.surface()
		);
	}
#endif
}

float Testbed::get_depth_from_renderbuffer(const CudaRenderBuffer& render_buffer, const vec2& uv) {
	if (!render_buffer.depth_buffer()) {
		return m_scale;
	}

	float depth;
	auto res = render_buffer.in_resolution();
	ivec2 depth_pixel = clamp(ivec2(uv * vec2(res)), 0, res - 1);

	CUDA_CHECK_THROW(
		hipMemcpy(&depth, render_buffer.depth_buffer() + depth_pixel.x + depth_pixel.y * res.x, sizeof(float), hipMemcpyDeviceToHost)
	);
	return depth;
}

vec3 Testbed::get_3d_pos_from_pixel(const CudaRenderBuffer& render_buffer, const vec2& pixel) {
	float depth = get_depth_from_renderbuffer(render_buffer, pixel / vec2(m_window_res));
	auto ray = pixel_to_ray_pinhole(
		0,
		ivec2(pixel),
		m_window_res,
		calc_focal_length(m_window_res, m_relative_focal_length, m_fov_axis, m_zoom),
		m_smoothed_camera,
		render_screen_center(m_screen_center)
	);
	return ray(depth);
}

void Testbed::autofocus() {
	float new_slice_plane_z = std::max(dot(view_dir(), m_autofocus_target - view_pos()), 0.1f) - m_scale;
	if (new_slice_plane_z != m_slice_plane_z) {
		m_slice_plane_z = new_slice_plane_z;
		if (m_aperture_size != 0.0f) {
			reset_accumulation();
		}
	}
}

Testbed::LevelStats compute_level_stats(const float* params, size_t n_params) {
	Testbed::LevelStats s = {};
	for (size_t i = 0; i < n_params; ++i) {
		float v = params[i];
		float av = fabsf(v);
		if (av < 0.00001f) {
			s.numzero++;
		} else {
			if (s.count == 0) {
				s.min = s.max = v;
			}
			s.count++;
			s.x += v;
			s.xsquared += v * v;
			s.min = min(s.min, v);
			s.max = max(s.max, v);
		}
	}
	return s;
}

Testbed::CudaDevice::CudaDevice(int id, bool is_primary) : m_id{id}, m_is_primary{is_primary} {
	auto guard = device_guard();
	m_stream = std::make_unique<StreamAndEvent>();
	m_data = std::make_unique<Data>();
	m_render_worker = std::make_unique<ThreadPool>(is_primary ? 0u : 1u);
}

ScopeGuard Testbed::CudaDevice::device_guard() {
	int prev_device = cuda_device();
	if (prev_device == m_id) {
		return {};
	}

	set_cuda_device(m_id);
	return ScopeGuard{[prev_device]() { set_cuda_device(prev_device); }};
}

void Testbed::sync_device(CudaRenderBuffer& render_buffer, Testbed::CudaDevice& device) {
	if (!device.dirty()) {
		return;
	}

	if (device.is_primary()) {
		device.data().hidden_area_mask = render_buffer.hidden_area_mask();
		device.set_dirty(false);
		return;
	}

	m_stream.signal(device.stream());

	int active_device = cuda_device();
	auto guard = device.device_guard();

	if (render_buffer.hidden_area_mask()) {
		auto ham = std::make_shared<Buffer2D<uint8_t>>(render_buffer.hidden_area_mask()->resolution());
		CUDA_CHECK_THROW(hipMemcpyPeerAsync(
			ham->data(), device.id(), render_buffer.hidden_area_mask()->data(), active_device, ham->bytes(), device.stream()
		));
		device.data().hidden_area_mask = ham;
	} else {
		device.data().hidden_area_mask = nullptr;
	}

	device.set_dirty(false);
	device.signal(m_stream.get());
}

// From https://stackoverflow.com/questions/20843271/passing-a-non-copyable-closure-object-to-stdfunction-parameter
template <class F> auto make_copyable_function(F&& f) {
	using dF = std::decay_t<F>;
	auto spf = std::make_shared<dF>(std::forward<F>(f));
	return [spf](auto&&... args) -> decltype(auto) { return (*spf)(decltype(args)(args)...); };
}

ScopeGuard Testbed::use_device(hipStream_t stream, CudaRenderBuffer& render_buffer, Testbed::CudaDevice& device) {
	device.wait_for(stream);

	if (device.is_primary()) {
		device.set_render_buffer_view(render_buffer.view());
		return ScopeGuard{[&device, stream]() {
			device.set_render_buffer_view({});
			device.signal(stream);
		}};
	}

	int active_device = cuda_device();
	auto guard = device.device_guard();

	size_t n_pixels = product(render_buffer.in_resolution());

	GPUMemoryArena::Allocation alloc;
	auto scratch = allocate_workspace_and_distribute<vec4, float>(device.stream(), &alloc, n_pixels, n_pixels);

	device.set_render_buffer_view({
		std::get<0>(scratch),
		std::get<1>(scratch),
		render_buffer.in_resolution(),
		render_buffer.spp(),
		device.data().hidden_area_mask,
	});

	return ScopeGuard{
		make_copyable_function([&render_buffer, &device, guard = std::move(guard), alloc = std::move(alloc), active_device, stream]() {
			// Copy device's render buffer's data onto the original render buffer
			CUDA_CHECK_THROW(hipMemcpyPeerAsync(
				render_buffer.frame_buffer(),
				active_device,
				device.render_buffer_view().frame_buffer,
				device.id(),
				product(render_buffer.in_resolution()) * sizeof(vec4),
				device.stream()
			));
			CUDA_CHECK_THROW(hipMemcpyPeerAsync(
				render_buffer.depth_buffer(),
				active_device,
				device.render_buffer_view().depth_buffer,
				device.id(),
				product(render_buffer.in_resolution()) * sizeof(float),
				device.stream()
			));

			device.set_render_buffer_view({});
			device.signal(stream);
		})
	};
}

void Testbed::set_all_devices_dirty() {
	for (auto& device : m_devices) {
		device.set_dirty(true);
	}
}

void Testbed::load_camera_path(const fs::path& path) { m_camera_path.load(path, mat4x3::identity()); }

bool Testbed::loop_animation() { return m_camera_path.loop; }

void Testbed::set_loop_animation(bool value) { m_camera_path.loop = value; }

} // namespace ngp
