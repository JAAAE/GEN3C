#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/** @file   python_api.cpp
 *  @author Thomas Müller & Alex Evans, NVIDIA
 */

#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/testbed.h>
#include <neural-graphics-primitives/thread_pool.h>

#include <json/json.hpp>

#include <pybind11/functional.h>
#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11_json/pybind11_json.hpp>
#include <tiny-cuda-nn/vec_pybind11.h>
#include <tinylogger/tinylogger.h>

#include <filesystem/path.h>

#ifdef NGP_GUI
#	include <imgui/imgui.h>
#	ifdef _WIN32
#		include <GL/gl3w.h>
#	else
#		include <GL/glew.h>
#	endif
#	include <GLFW/glfw3.h>
#endif

using namespace nlohmann;
namespace py = pybind11;

namespace ngp {

// Returns RGBA and depth buffers
std::pair<py::array_t<float>, py::array_t<float>>
	Testbed::render_to_cpu(int width, int height, int spp, bool linear, float start_time, float end_time, float fps, float shutter_fraction) {
	m_windowless_render_surface.resize({width, height});
	m_windowless_render_surface.reset_accumulation();

	if (end_time < 0.f) {
		end_time = start_time;
	}

	bool path_animation_enabled = start_time >= 0.f;
	if (!path_animation_enabled) { // the old code disabled camera smoothing for non-path renders; so we preserve that behaviour
		m_smoothed_camera = m_camera;
	}

	// this rendering code assumes that the intra-frame camera motion starts from m_smoothed_camera (ie where we left off) to allow for EMA
	// camera smoothing. in the case of a camera path animation, at the very start of the animation, we have yet to initialize
	// smoothed_camera to something sensible
	// - it will just be the default boot position. oops!
	// that led to the first frame having a crazy streak from the default camera position to the start of the path.
	// so we detect that case and explicitly force the current matrix to the start of the path
	if (start_time == 0.f) {
		set_camera_from_time(start_time);
		m_smoothed_camera = m_camera;
	}

	auto start_cam_matrix = m_smoothed_camera;

	// now set up the end-of-frame camera matrix if we are moving along a path
	if (path_animation_enabled) {
		set_camera_from_time(end_time);
		apply_camera_smoothing(1000.f / fps);
	}

	auto end_cam_matrix = m_smoothed_camera;
	auto prev_camera_matrix = m_smoothed_camera;

	for (int i = 0; i < spp; ++i) {
		float start_alpha = ((float)i) / (float)spp * shutter_fraction;
		float end_alpha = ((float)i + 1.0f) / (float)spp * shutter_fraction;

		auto sample_start_cam_matrix = start_cam_matrix;
		auto sample_end_cam_matrix = camera_log_lerp(start_cam_matrix, end_cam_matrix, shutter_fraction);
		if (i == 0) {
			prev_camera_matrix = sample_start_cam_matrix;
		}

		if (path_animation_enabled) {
			set_camera_from_time(start_time + (end_time - start_time) * (start_alpha + end_alpha) / 2.0f);
			m_smoothed_camera = m_camera;
		}

		if (m_autofocus) {
			autofocus();
		}

		render_frame(
			m_stream.get(),
			sample_start_cam_matrix,
			sample_end_cam_matrix,
			prev_camera_matrix,
			m_screen_center,
			m_relative_focal_length,
			{}, // foveation
			{}, // prev foveation
			{}, // lens
			m_visualized_dimension,
			m_windowless_render_surface,
			!linear
		);
		prev_camera_matrix = sample_start_cam_matrix;
	}

	// For cam smoothing when rendering the next frame.
	m_smoothed_camera = end_cam_matrix;

	py::array_t<float> result_rgba({height, width, 4});
	py::buffer_info buf_rgba = result_rgba.request();

	py::array_t<float> result_depth({height, width});
	py::buffer_info buf_depth = result_depth.request();

	CUDA_CHECK_THROW(hipMemcpy2DFromArray(
		buf_rgba.ptr, width * sizeof(float) * 4, m_windowless_render_surface.surface_provider().array(), 0, 0, width * sizeof(float) * 4, height, hipMemcpyDeviceToHost
	));

	CUDA_CHECK_THROW(
		hipMemcpy(buf_depth.ptr, m_windowless_render_surface.depth_buffer(), height * width * sizeof(float), hipMemcpyDeviceToHost)
	);

	return {result_rgba, result_depth};
}

py::array_t<float> Testbed::render_to_cpu_rgba(
	int width, int height, int spp, bool linear, float start_time, float end_time, float fps, float shutter_fraction
) {
	return render_to_cpu(width, height, spp, linear, start_time, end_time, fps, shutter_fraction).first;
}

py::array_t<float> Testbed::view(bool linear, size_t view_idx) const {
	if (m_views.size() <= view_idx) {
		throw std::runtime_error{fmt::format("View #{} does not exist.", view_idx)};
	}

	auto& view = m_views.at(view_idx);
	auto& render_buffer = *view.render_buffer;

	auto res = render_buffer.out_resolution();

	py::array_t<float> result({res.y, res.x, 4});
	py::buffer_info buf = result.request();
	float* data = (float*)buf.ptr;

	CUDA_CHECK_THROW(hipMemcpy2DFromArray(
		data, res.x * sizeof(float) * 4, render_buffer.surface_provider().array(), 0, 0, res.x * sizeof(float) * 4, res.y, hipMemcpyDeviceToHost
	));

	if (linear) {
		ThreadPool{}.parallel_for<size_t>(0, res.y, [&](size_t y) {
			size_t base = y * res.x;
			for (uint32_t x = 0; x < res.x; ++x) {
				size_t px = base + x;
				data[px * 4 + 0] = srgb_to_linear(data[px * 4 + 0]);
				data[px * 4 + 1] = srgb_to_linear(data[px * 4 + 1]);
				data[px * 4 + 2] = srgb_to_linear(data[px * 4 + 2]);
			}
		});
	}

	return result;
}

std::pair<py::array_t<float>, py::array_t<uint32_t>>
	Testbed::reproject(const mat4x3& src, const py::array_t<float>& src_img, const py::array_t<float>& src_depth, const mat4x3& dst) {

	py::buffer_info src_img_buf = src_img.request();
	py::buffer_info src_depth_buf = src_depth.request();

	if (src_img_buf.ndim != 3) {
		throw std::runtime_error{"src image should be (H,W,C) where C=4"};
	}

	if (src_img_buf.shape[2] != 4) {
		throw std::runtime_error{"src image should be (H,W,C) where C=4"};
	}

	if (src_depth_buf.ndim != 2) {
		throw std::runtime_error{"src depth should be (H,W)"};
	}

	if (src_img_buf.shape[0] != src_depth_buf.shape[0] || src_img_buf.shape[1] != src_depth_buf.shape[1]) {
		throw std::runtime_error{"image and depth dimensions don't match"};
	}

	const ivec2 src_res = {(int)src_img_buf.shape[1], (int)src_img_buf.shape[0]};
	const ivec2 dst_res = src_res; // For now

	auto src_render_buffer = std::make_shared<CudaRenderBuffer>(std::make_shared<CudaSurface2D>());
	src_render_buffer->resize(src_res);

	auto dst_render_buffer = std::make_shared<CudaRenderBuffer>(std::make_shared<CudaSurface2D>());
	dst_render_buffer->resize(dst_res);

	View src_view, dst_view;

	src_view.camera0 = src_view.camera1 = src_view.prev_camera = src;
	src_view.device = &primary_device();
	src_view.foveation = src_view.prev_foveation = {};
	src_view.screen_center = vec2(0.5f);
	src_view.full_resolution = src_res;
	src_view.visualized_dimension = -1;
	src_view.relative_focal_length = m_relative_focal_length;
	src_view.render_buffer = src_render_buffer;

	dst_view.camera0 = dst_view.camera1 = dst_view.prev_camera = dst;
	dst_view.device = &primary_device();
	dst_view.foveation = dst_view.prev_foveation = {};
	dst_view.screen_center = vec2(0.5f);
	dst_view.full_resolution = dst_res;
	dst_view.visualized_dimension = -1;
	dst_view.relative_focal_length = m_relative_focal_length;
	dst_view.render_buffer = dst_render_buffer;

	CUDA_CHECK_THROW(hipMemcpyAsync(
		src_render_buffer->frame_buffer(), src_img_buf.ptr, product(src_res) * sizeof(float) * 4, hipMemcpyHostToDevice, m_stream.get()
	));
	CUDA_CHECK_THROW(hipMemcpyAsync(
		src_render_buffer->depth_buffer(), src_depth_buf.ptr, product(src_res) * sizeof(float), hipMemcpyHostToDevice, m_stream.get()
	));

	std::vector<const View*> src_views = {&src_view};
	reproject_views(src_views, dst_view);

	py::array_t<float> result_rgba({dst_res.y, dst_res.x, 4});
	py::buffer_info buf_rgba = result_rgba.request();

	py::array_t<uint32_t> result_idx({dst_res.y, dst_res.x});
	py::buffer_info buf_idx = result_idx.request();

	CUDA_CHECK_THROW(hipMemcpyAsync(
		buf_rgba.ptr, dst_render_buffer->frame_buffer(), product(dst_res) * sizeof(float) * 4, hipMemcpyDeviceToHost, m_stream.get()
	));

	auto idx_buffer = GPUImage<uint32_t>(dst_res, m_stream.get());

	parallel_for_gpu(
		m_stream.get(),
		idx_buffer.n_elements(),
		[out = idx_buffer.view(), in = dst_view.index_field.view(), src_width = src_res.x, dst_width = dst_res.x] __device__(size_t i) {
			ivec2 idx = ivec2(i % dst_width, i / dst_width);
			ivec2 src_idx = in(idx.y, idx.x).px;
			out(idx.y, idx.x) = src_idx.x + src_idx.y * src_width;
		}
	);

	CUDA_CHECK_THROW(
		hipMemcpyAsync(buf_idx.ptr, idx_buffer.data(), product(dst_res) * sizeof(uint32_t), hipMemcpyDeviceToHost, m_stream.get())
	);

	return {result_rgba, result_idx};
}

uint32_t Testbed::add_src_view(
	mat4x3 camera_to_world, float fx, float fy, float cx, float cy, Lens lens, pybind11::array_t<float> img, pybind11::array_t<float> depth, float timestamp, bool is_srgb
) {
	py::buffer_info src_img_buf = img.request();
	py::buffer_info src_depth_buf = depth.request();

	if (src_img_buf.ndim != 3) {
		throw std::runtime_error{"src image should be (H,W,C) where C=4"};
	}

	if (src_img_buf.shape[2] != 4) {
		throw std::runtime_error{"src image should be (H,W,C) where C=4"};
	}

	if (src_depth_buf.ndim != 2) {
		throw std::runtime_error{"src depth should be (H,W)"};
	}

	if (src_img_buf.shape[0] != src_depth_buf.shape[0] || src_img_buf.shape[1] != src_depth_buf.shape[1]) {
		throw std::runtime_error{"image and depth dimensions don't match"};
	}

	const ivec2 src_res = {(int)src_img_buf.shape[1], (int)src_img_buf.shape[0]};

	static uint32_t id = 0;

	m_reproject_src_views.emplace_back();
	if (m_reproject_max_src_view_count > 0 && m_reproject_src_views.size() > (size_t)m_reproject_max_src_view_count) {
		m_reproject_src_views.pop_front();
	}

	auto& src_view = m_reproject_src_views.back();
	src_view.uid = id++;
	src_view.camera0 = src_view.camera1 = src_view.prev_camera = camera_to_world;
	src_view.device = &primary_device();
	src_view.foveation = src_view.prev_foveation = {};
	src_view.screen_center = vec2(cx, cy);
	src_view.full_resolution = src_res;
	src_view.visualized_dimension = -1;
	src_view.relative_focal_length = vec2(fx, fy) / (float)src_res[m_fov_axis];
	src_view.render_buffer = std::make_shared<CudaRenderBuffer>(std::make_shared<CudaSurface2D>());
	src_view.render_buffer->resize(src_res);
	src_view.lens = lens;

	CUDA_CHECK_THROW(hipMemcpyAsync(
		src_view.render_buffer->frame_buffer(), src_img_buf.ptr, product(src_res) * sizeof(float) * 4, hipMemcpyHostToDevice, m_stream.get()
	));
	CUDA_CHECK_THROW(hipMemcpyAsync(
		src_view.render_buffer->depth_buffer(), src_depth_buf.ptr, product(src_res) * sizeof(float), hipMemcpyHostToDevice, m_stream.get()
	));

	if (is_srgb) {
		// Convert from sRGB to linear on the GPU directly
		parallel_for_gpu(
			m_stream.get(),
			product(src_res) * 4,
			[values = (float *) src_view.render_buffer->frame_buffer()] __device__(size_t i) {
				if ((i % 4) == 3) {
					// Don't linearize the alpha channel
					return;
				}
				values[i] = srgb_to_linear(values[i]);
			}
		);
	}

	return src_view.uid;
}


pybind11::array_t<uint32_t> Testbed::src_view_ids() const {
	py::array_t<uint32_t> result({(int)m_reproject_src_views.size()});
	py::buffer_info buf = result.request();
	uint32_t* data = (uint32_t*)buf.ptr;
	for (size_t i = 0; i < m_reproject_src_views.size(); ++i) {
		data[i] = m_reproject_src_views[i].uid;
	}
	return result;
}

#ifdef NGP_GUI
py::array_t<float> Testbed::screenshot(bool linear, bool front_buffer) const {
	std::vector<float> tmp(product(m_window_res) * 4);
	glReadBuffer(front_buffer ? GL_FRONT : GL_BACK);
	glReadPixels(0, 0, m_window_res.x, m_window_res.y, GL_RGBA, GL_FLOAT, tmp.data());

	py::array_t<float> result({m_window_res.y, m_window_res.x, 4});
	py::buffer_info buf = result.request();
	float* data = (float*)buf.ptr;

	// Linear, alpha premultiplied, Y flipped
	ThreadPool{}.parallel_for<size_t>(0, m_window_res.y, [&](size_t y) {
		size_t base = y * m_window_res.x;
		size_t base_reverse = (m_window_res.y - y - 1) * m_window_res.x;
		for (uint32_t x = 0; x < m_window_res.x; ++x) {
			size_t px = base + x;
			size_t px_reverse = base_reverse + x;
			data[px_reverse * 4 + 0] = linear ? srgb_to_linear(tmp[px * 4 + 0]) : tmp[px * 4 + 0];
			data[px_reverse * 4 + 1] = linear ? srgb_to_linear(tmp[px * 4 + 1]) : tmp[px * 4 + 1];
			data[px_reverse * 4 + 2] = linear ? srgb_to_linear(tmp[px * 4 + 2]) : tmp[px * 4 + 2];
			data[px_reverse * 4 + 3] = tmp[px * 4 + 3];
		}
	});

	return result;
}
#endif

PYBIND11_MODULE(pyngp, m) {
	m.doc() = "Gen3C GUI";

	m.def("free_temporary_memory", &free_all_gpu_memory_arenas);

	py::enum_<ETestbedMode>(m, "TestbedMode")
		.value("Gen3c", ETestbedMode::Gen3c)
		.value("None", ETestbedMode::None)
		.export_values();

	m.def("mode_from_scene", &mode_from_scene);
	m.def("mode_from_string", &mode_from_string);

	py::enum_<EGroundTruthRenderMode>(m, "GroundTruthRenderMode")
		.value("Shade", EGroundTruthRenderMode::Shade)
		.value("Depth", EGroundTruthRenderMode::Depth)
		.export_values();

	py::enum_<ERenderMode>(m, "RenderMode")
		.value("AO", ERenderMode::AO)
		.value("Shade", ERenderMode::Shade)
		.value("Normals", ERenderMode::Normals)
		.value("Positions", ERenderMode::Positions)
		.value("Depth", ERenderMode::Depth)
		.value("Distortion", ERenderMode::Distortion)
		.value("Cost", ERenderMode::Cost)
		.value("Slice", ERenderMode::Slice)
		.export_values();

	py::enum_<ERandomMode>(m, "RandomMode")
		.value("Random", ERandomMode::Random)
		.value("Halton", ERandomMode::Halton)
		.value("Sobol", ERandomMode::Sobol)
		.value("Stratified", ERandomMode::Stratified)
		.export_values();

	py::enum_<ELossType>(m, "LossType")
		.value("L2", ELossType::L2)
		.value("L1", ELossType::L1)
		.value("Mape", ELossType::Mape)
		.value("Smape", ELossType::Smape)
		.value("Huber", ELossType::Huber)
		// Legacy: we used to refer to the Huber loss
		// (L2 near zero, L1 further away) as "SmoothL1".
		.value("SmoothL1", ELossType::Huber)
		.value("LogL1", ELossType::LogL1)
		.value("RelativeL2", ELossType::RelativeL2)
		.export_values();

	py::enum_<ESDFGroundTruthMode>(m, "SDFGroundTruthMode")
		.value("RaytracedMesh", ESDFGroundTruthMode::RaytracedMesh)
		.value("SpheretracedMesh", ESDFGroundTruthMode::SpheretracedMesh)
		.value("SDFBricks", ESDFGroundTruthMode::SDFBricks)
		.export_values();

	py::enum_<EMeshSdfMode>(m, "MeshSdfMode")
		.value("Watertight", EMeshSdfMode::Watertight)
		.value("Raystab", EMeshSdfMode::Raystab)
		.value("PathEscape", EMeshSdfMode::PathEscape)
		.export_values();

	py::enum_<EColorSpace>(m, "ColorSpace").value("Linear", EColorSpace::Linear).value("SRGB", EColorSpace::SRGB).export_values();

	py::enum_<ETonemapCurve>(m, "TonemapCurve")
		.value("Identity", ETonemapCurve::Identity)
		.value("ACES", ETonemapCurve::ACES)
		.value("Hable", ETonemapCurve::Hable)
		.value("Reinhard", ETonemapCurve::Reinhard)
		.export_values();

	py::enum_<ELensMode>(m, "LensMode")
		.value("Perspective", ELensMode::Perspective)
		.value("OpenCV", ELensMode::OpenCV)
		.value("FTheta", ELensMode::FTheta)
		.value("LatLong", ELensMode::LatLong)
		.value("OpenCVFisheye", ELensMode::OpenCVFisheye)
		.value("Equirectangular", ELensMode::Equirectangular)
		.value("Orthographic", ELensMode::Orthographic)
		.export_values();


	py::class_<BoundingBox>(m, "BoundingBox")
		.def(py::init<>())
		.def(py::init<const vec3&, const vec3&>())
		.def("center", &BoundingBox::center)
		.def("contains", &BoundingBox::contains)
		.def("diag", &BoundingBox::diag)
		.def("distance", &BoundingBox::distance)
		.def("distance_sq", &BoundingBox::distance_sq)
		.def("enlarge", py::overload_cast<const vec3&>(&BoundingBox::enlarge))
		.def("enlarge", py::overload_cast<const BoundingBox&>(&BoundingBox::enlarge))
		.def("get_vertices", &BoundingBox::get_vertices)
		.def("inflate", &BoundingBox::inflate)
		.def("intersection", &BoundingBox::intersection)
		.def("intersects", py::overload_cast<const BoundingBox&>(&BoundingBox::intersects, py::const_))
		.def("ray_intersect", &BoundingBox::ray_intersect)
		.def("relative_pos", &BoundingBox::relative_pos)
		.def("signed_distance", &BoundingBox::signed_distance)
		.def_readwrite("min", &BoundingBox::min)
		.def_readwrite("max", &BoundingBox::max);

	py::class_<Lens> lens(m, "Lens");
	lens.def(py::init<>()).def_readwrite("mode", &Lens::mode).def_property_readonly("params", [](py::object& obj) {
		Lens& o = obj.cast<Lens&>();
		return py::array{sizeof(o.params) / sizeof(o.params[0]), o.params, obj};
	});

	m.def("fov_to_focal_length", py::overload_cast<int, float>(&ngp::fov_to_focal_length),
		  py::arg("resolution"), py::arg("degrees"))
	 .def("fov_to_focal_length", py::overload_cast<const ivec2&, const vec2&>(&fov_to_focal_length),
		  py::arg("resolution"), py::arg("degrees"))
	 .def("focal_length_to_fov", py::overload_cast<int, float>(&ngp::focal_length_to_fov),
		  py::arg("resolution"), py::arg("focal_length"))
	 .def("focal_length_to_fov", py::overload_cast<const ivec2&, const vec2&>(&ngp::focal_length_to_fov),
		  py::arg("resolution"), py::arg("focal_length"))
	 .def("relative_focal_length_to_fov", &ngp::relative_focal_length_to_fov,
		  py::arg("rel_focal_length"));

	py::class_<fs::path>(m, "path").def(py::init<>()).def(py::init<const std::string&>());

	py::implicitly_convertible<std::string, fs::path>();

	py::class_<Testbed> testbed(m, "Testbed");
	testbed.def(py::init<ETestbedMode>(), py::arg("mode") = ETestbedMode::None)
		.def_readonly("mode", &Testbed::m_testbed_mode)
		// General control
		.def(
			"init_window",
			&Testbed::init_window,
			"Init a GLFW window that shows real-time progress and a GUI. 'second_window' creates a second copy of the output in its own window.",
			py::arg("width"),
			py::arg("height"),
			py::arg("hidden") = false,
			py::arg("second_window") = false
		)
		.def("destroy_window", &Testbed::destroy_window, "Destroy the window again.")
		.def(
			"init_vr",
			&Testbed::init_vr,
			"Init rendering to a connected and active VR headset. Requires a window to have been previously created via `init_window`."
		)
		.def(
			"view",
			&Testbed::view,
			"Outputs the currently displayed image by a given view (0 by default).",
			py::arg("linear") = true,
			py::arg("view") = 0
		)
		.def("view_camera", &Testbed::view_camera, "Outputs the current camera matrix of a given view (0 by default).", py::arg("view") = 0)
		.def(
			"add_src_view",
			&Testbed::add_src_view,
			"Adds a source view to the pool of views for reprojection.",
			py::arg("camera_to_world"),
			py::arg("fx"),
			py::arg("fy"),
			py::arg("cx"),
			py::arg("cy"),
			py::arg("img"),
			py::arg("depth"),
			py::arg("lens"),
			py::arg("timestamp"),
			py::arg("is_srgb") = false
		)
		.def("src_view_ids", &Testbed::src_view_ids, "Returns the IDs of all source views currently registered.")
		.def("clear_src_views", &Testbed::clear_src_views, "Remove all views from the pool of views for reprojection.")
#ifdef NGP_GUI
		.def_readwrite("keyboard_event_callback", &Testbed::m_keyboard_event_callback)
		.def_readwrite("file_drop_callback", &Testbed::m_file_drop_callback)
		.def("is_key_pressed", [](py::object& obj, int key) { return ImGui::IsKeyPressed(key); })
		.def("is_key_down", [](py::object& obj, int key) { return ImGui::IsKeyDown(key); })
		.def("is_alt_down", [](py::object& obj) { return ImGui::GetIO().KeyMods & ImGuiKeyModFlags_Alt; })
		.def("is_ctrl_down", [](py::object& obj) { return ImGui::GetIO().KeyMods & ImGuiKeyModFlags_Ctrl; })
		.def("is_shift_down", [](py::object& obj) { return ImGui::GetIO().KeyMods & ImGuiKeyModFlags_Shift; })
		.def("is_super_down", [](py::object& obj) { return ImGui::GetIO().KeyMods & ImGuiKeyModFlags_Super; })
		.def(
			"screenshot",
			&Testbed::screenshot,
			"Takes a screenshot of the current window contents.",
			py::arg("linear") = true,
			py::arg("front_buffer") = true
		)
		.def_readwrite("vr_use_hidden_area_mask", &Testbed::m_vr_use_hidden_area_mask)
		.def_readwrite("vr_use_depth_reproject", &Testbed::m_vr_use_depth_reproject)
#endif
		.def("want_repl", &Testbed::want_repl, "returns true if the user clicked the 'I want a repl' button")
		.def(
			"frame", &Testbed::frame, py::call_guard<py::gil_scoped_release>(), "Process a single frame. Renders if a window was previously created."
		)
		.def(
			"render",
			&Testbed::render_to_cpu_rgba,
			"Renders an image at the requested resolution. Does not require a window.",
			py::arg("width") = 1920,
			py::arg("height") = 1080,
			py::arg("spp") = 1,
			py::arg("linear") = true,
			py::arg("start_t") = -1.f,
			py::arg("end_t") = -1.f,
			py::arg("fps") = 30.f,
			py::arg("shutter_fraction") = 1.0f
		)
		.def(
			"render_with_depth",
			&Testbed::render_to_cpu,
			"Renders an image at the requested resolution. Does not require a window.",
			py::arg("width") = 1920,
			py::arg("height") = 1080,
			py::arg("spp") = 1,
			py::arg("linear") = true,
			py::arg("start_t") = -1.f,
			py::arg("end_t") = -1.f,
			py::arg("fps") = 30.f,
			py::arg("shutter_fraction") = 1.0f
		)
		.def("reproject", &Testbed::reproject, "Reprojects an RGBA + depth image from a known camera view to another camera view.")
		.def("reset_camera", &Testbed::reset_camera, "Reset camera to default state.")
		.def(
			"reset_accumulation",
			&Testbed::reset_accumulation,
			"Reset rendering accumulation.",
			py::arg("due_to_camera_movement") = false,
			py::arg("immediate_redraw") = true,
			py::arg("reset_pip") = false
		)
		.def("load_camera_path", &Testbed::load_camera_path, py::arg("path"), "Load a camera path")
		.def(
			"load_file",
			&Testbed::load_file,
			py::arg("path"),
			"Load a file and automatically determine how to handle it. Can be a snapshot, dataset, network config, or camera path."
		)
		.def_property("loop_animation", &Testbed::loop_animation, &Testbed::set_loop_animation)
		// Interesting members.
		.def_readwrite("reproject_min_t", &Testbed::m_reproject_min_t)
		.def_readwrite("reproject_step_factor", &Testbed::m_reproject_step_factor)
		.def_readwrite("reproject_parallax", &Testbed::m_reproject_parallax)
		.def_readwrite("reproject_second_view", &Testbed::m_reproject_enable)
		.def_readwrite("reproject_enable", &Testbed::m_reproject_enable)
		.def_readwrite("reproject_visualize_src_views", &Testbed::m_reproject_visualize_src_views)
		.def_readwrite("reproject_min_src_view_index", &Testbed::m_reproject_min_src_view_index)
		.def_readwrite("reproject_max_src_view_index", &Testbed::m_reproject_max_src_view_index)
		.def_readwrite("reproject_max_src_view_count", &Testbed::m_reproject_max_src_view_count)
		.def("reproject_src_views_count", [](const Testbed& testbed) { return testbed.m_reproject_src_views.size(); })
		.def_readwrite("reproject_reuse_last_frame", &Testbed::m_reproject_reuse_last_frame)
		.def("init_camera_path_from_reproject_src_cameras", &Testbed::init_camera_path_from_reproject_src_cameras)
		.def_readwrite("pm_enable", &Testbed::m_pm_enable)
		.def_readwrite("dynamic_res", &Testbed::m_dynamic_res)
		.def_readwrite("dynamic_res_target_fps", &Testbed::m_dynamic_res_target_fps)
		.def_readwrite("fixed_res_factor", &Testbed::m_fixed_res_factor)
		.def_readwrite("background_color", &Testbed::m_background_color)
		.def_readwrite("render_transparency_as_checkerboard", &Testbed::m_render_transparency_as_checkerboard)
		.def_readwrite("render_groundtruth", &Testbed::m_render_ground_truth)
		.def_readwrite("render_ground_truth", &Testbed::m_render_ground_truth)
		.def_readwrite("groundtruth_render_mode", &Testbed::m_ground_truth_render_mode)
		.def_readwrite("render_mode", &Testbed::m_render_mode)
		.def_readwrite("render_near_distance", &Testbed::m_render_near_distance)
		.def_readwrite("slice_plane_z", &Testbed::m_slice_plane_z)
		.def_readwrite("dof", &Testbed::m_aperture_size)
		.def_readwrite("aperture_size", &Testbed::m_aperture_size)
		.def_readwrite("autofocus", &Testbed::m_autofocus)
		.def_readwrite("autofocus_target", &Testbed::m_autofocus_target)
		.def_readwrite("camera_path", &Testbed::m_camera_path)
		.def_readwrite("record_camera_path", &Testbed::m_record_camera_path)
		.def_readwrite("floor_enable", &Testbed::m_floor_enable)
		.def_readwrite("exposure", &Testbed::m_exposure)
		.def_property("scale", &Testbed::scale, &Testbed::set_scale)
		.def_readonly("bounding_radius", &Testbed::m_bounding_radius)
		.def_readwrite("render_aabb", &Testbed::m_render_aabb)
		.def_readwrite("render_aabb_to_local", &Testbed::m_render_aabb_to_local)
		.def_readwrite("is_rendering", &Testbed::m_render)
		.def_readwrite("aabb", &Testbed::m_aabb)
		.def_readwrite("raw_aabb", &Testbed::m_raw_aabb)
		.def_property("fov", &Testbed::fov, &Testbed::set_fov)
		.def_property("fov_xy", &Testbed::fov_xy, &Testbed::set_fov_xy)
		.def_readwrite("fov_axis", &Testbed::m_fov_axis)
		.def_readwrite("relative_focal_length", &Testbed::m_relative_focal_length)
		.def_readwrite("zoom", &Testbed::m_zoom)
		.def_readwrite("screen_center", &Testbed::m_screen_center)
		.def_readwrite("camera_matrix", &Testbed::m_camera)
		.def_readwrite("up_dir", &Testbed::m_up_dir)
		.def_readwrite("sun_dir", &Testbed::m_sun_dir)
		.def_readwrite("default_camera", &Testbed::m_default_camera)
		.def_property("look_at", &Testbed::look_at, &Testbed::set_look_at)
		.def_property("view_dir", &Testbed::view_dir, &Testbed::set_view_dir)
		.def_readwrite("camera_smoothing", &Testbed::m_camera_smoothing)
		.def_readwrite("render_with_lens_distortion", &Testbed::m_render_with_lens_distortion)
		.def_readwrite("render_lens", &Testbed::m_render_lens)
		.def_property(
			"display_gui",
			[](py::object& obj) { return obj.cast<Testbed&>().m_imgui.mode == Testbed::ImGuiMode::Enabled; },
			[](const py::object& obj, bool value) {
				obj.cast<Testbed&>().m_imgui.mode = value ? Testbed::ImGuiMode::Enabled : Testbed::ImGuiMode::Disabled;
			}
		)
		.def_property(
			"video_path",
			[](Testbed& obj) { return obj.m_imgui.video_path; },
			[](Testbed& obj, const std::string& value) {
				if (value.size() > Testbed::ImGuiVars::MAX_PATH_LEN)
					throw std::runtime_error{"Video path is too long."};
				strcpy(obj.m_imgui.video_path, value.c_str());
			}
		)
		.def_readwrite("visualize_unit_cube", &Testbed::m_visualize_unit_cube)
		.def_readwrite("snap_to_pixel_centers", &Testbed::m_snap_to_pixel_centers)
		.def_readwrite("parallax_shift", &Testbed::m_parallax_shift)
		.def_readwrite("color_space", &Testbed::m_color_space)
		.def_readwrite("tonemap_curve", &Testbed::m_tonemap_curve)
		.def_property(
			"dlss",
			[](py::object& obj) { return obj.cast<Testbed&>().m_dlss; },
			[](const py::object& obj, bool value) {
				if (value && !obj.cast<Testbed&>().m_dlss_provider) {
					if (obj.cast<Testbed&>().m_render_window) {
						throw std::runtime_error{"DLSS not supported."};
					} else {
						throw std::runtime_error{"DLSS requires a Window to be initialized via `init_window`."};
					}
				}

				obj.cast<Testbed&>().m_dlss = value;
			}
		)
		.def_readwrite("dlss_sharpening", &Testbed::m_dlss_sharpening)
		.def_property(
			"root_dir",
			[](py::object& obj) { return obj.cast<Testbed&>().root_dir().str(); },
			[](const py::object& obj, const std::string& value) { obj.cast<Testbed&>().set_root_dir(value); }
		);

	py::enum_<EGen3cCameraSource>(m, "Gen3cCameraSource")
		.value("Fake", EGen3cCameraSource::Fake)
		.value("Viewpoint", EGen3cCameraSource::Viewpoint)
		.value("Authored", EGen3cCameraSource::Authored);

	testbed
		.def(
			"set_gen3c_cb",
			[](Testbed& testbed, const Testbed::gen3c_cb_t& cb) {
				// testbed.m_gen3c_cb.reset(cb);
				testbed.m_gen3c_cb = cb;
			}
		)
		.def_readwrite("gen3c_info", &Testbed::m_gen3c_info)
		.def_readwrite("gen3c_seed_path", &Testbed::m_gen3c_seed_path)
		.def_readwrite("gen3c_auto_inference", &Testbed::m_gen3c_auto_inference)
		.def_readwrite("gen3c_camera_source", &Testbed::m_gen3c_camera_source)
		.def_readwrite("gen3c_translation_speed", &Testbed::m_gen3c_translation_speed)
		.def_readwrite("gen3c_rotation_speed", &Testbed::m_gen3c_rotation_speed)
		.def_readwrite("gen3c_inference_info", &Testbed::m_gen3c_inference_info)
		.def_readwrite("gen3c_seeding_progress", &Testbed::m_gen3c_seeding_progress)
		.def_readwrite("gen3c_inference_progress", &Testbed::m_gen3c_inference_progress)
		.def_readwrite("gen3c_inference_is_connected", &Testbed::m_gen3c_inference_is_connected)
		.def_readwrite("gen3c_render_with_gen3c", &Testbed::m_gen3c_render_with_gen3c)
		// Output
		.def_readwrite("gen3c_save_frames", &Testbed::m_gen3c_save_frames)
		.def_readwrite("gen3c_display_frames", &Testbed::m_gen3c_display_frames)
		.def_readwrite("gen3c_output_dir", &Testbed::m_gen3c_output_dir)
		.def_readwrite("gen3c_show_cache_renderings", &Testbed::m_gen3c_show_cache_renderings);

	py::class_<CameraKeyframe>(m, "CameraKeyframe")
		.def(py::init<>())
		.def(
			py::init<const quat&, const vec3&, float, float>(),
			py::arg("r"),
			py::arg("t"),
			py::arg("fov"),
			py::arg("timestamp")
		)
		.def(
			py::init<const mat4x3&, float, float>(),
			py::arg("m"),
			py::arg("fov"),
			py::arg("timestamp")
		)
		.def_readwrite("R", &CameraKeyframe::R)
		.def_readwrite("T", &CameraKeyframe::T)
		.def_readwrite("fov", &CameraKeyframe::fov)
		.def_readwrite("timestamp", &CameraKeyframe::timestamp)
		.def("m", &CameraKeyframe::m)
		.def("from_m", &CameraKeyframe::from_m, py::arg("rv"))
		.def("same_pos_as", &CameraKeyframe::same_pos_as, py::arg("rhs"));

	py::enum_<EEditingKernel>(m, "EditingKernel")
		.value("None", EEditingKernel::None)
		.value("Gaussian", EEditingKernel::Gaussian)
		.value("Quartic", EEditingKernel::Quartic)
		.value("Hat", EEditingKernel::Hat)
		.value("Box", EEditingKernel::Box);

	py::class_<CameraPath::RenderSettings>(m, "CameraPathRenderSettings")
		.def_readwrite("resolution", &CameraPath::RenderSettings::resolution)
		.def_readwrite("spp", &CameraPath::RenderSettings::spp)
		.def_readwrite("fps", &CameraPath::RenderSettings::fps)
		.def_readwrite("shutter_fraction", &CameraPath::RenderSettings::shutter_fraction)
		.def_readwrite("quality", &CameraPath::RenderSettings::quality);

	py::class_<CameraPath::Pos>(m, "CameraPathPos").def_readwrite("kfidx", &CameraPath::Pos::kfidx).def_readwrite("t", &CameraPath::Pos::t);

	py::class_<CameraPath>(m, "CameraPath")
		.def_readwrite("keyframes", &CameraPath::keyframes)
		.def_readwrite("update_cam_from_path", &CameraPath::update_cam_from_path)
		.def_readwrite("play_time", &CameraPath::play_time)
		.def_readwrite("auto_play_speed", &CameraPath::auto_play_speed)
		.def_readwrite("default_duration_seconds", &CameraPath::default_duration_seconds)
		.def_readwrite("loop", &CameraPath::loop)
		.def_readwrite("keyframe_subsampling", &CameraPath::keyframe_subsampling)
		.def_property("duration_seconds", &CameraPath::duration_seconds, &CameraPath::set_duration_seconds)
		.def_readwrite("editing_kernel_type", &CameraPath::editing_kernel_type)
		.def_readwrite("editing_kernel_radius", &CameraPath::editing_kernel_radius)
		.def_readwrite("spline_order", &CameraPath::spline_order)
		.def_readwrite("render_settings", &CameraPath::render_settings)
		.def_readwrite("rendering", &CameraPath::rendering)
		.def_readwrite("render_frame_idx", &CameraPath::render_frame_idx)
		.def_readwrite("render_start_time", &CameraPath::render_start_time)
		.def_readwrite("render_frame_end_camera", &CameraPath::render_frame_end_camera)
		.def("clear", &CameraPath::clear)
		.def("has_valid_timestamps", &CameraPath::has_valid_timestamps)
		.def("make_keyframe_timestamps_equidistant", &CameraPath::make_keyframe_timestamps_equidistant)
		.def("sanitize_keyframes", &CameraPath::sanitize_keyframes)
		.def("get_pos", &CameraPath::get_pos, py::arg("playtime"))
		.def("get_playtime", &CameraPath::get_playtime, py::arg("i"))
		.def("get_keyframe", &CameraPath::get_keyframe, py::arg("i"))
		.def("eval_camera_path", &CameraPath::eval_camera_path, py::arg("t"))
		.def("save", &CameraPath::save, py::arg("path"))
		.def("load", &CameraPath::load, py::arg("path"), py::arg("first_xform"))
		.def(
			"add_camera",
			&CameraPath::add_camera,
			py::arg("camera"),
			py::arg("fov"),
			py::arg("timestamp")
		);

	// Minimal logging framework (tlog)
	// https://github.com/Tom94/tinylogger/
	py::module_ tlog = m.def_submodule("tlog", "Tiny logging framework");
	tlog.def("none", [](const std::string &s) { tlog::none() << s; })
		.def("info", [](const std::string &s) { tlog::info() << s; })
		.def("debug", [](const std::string &s) { tlog::debug() << s; })
		.def("warning", [](const std::string &s) { tlog::warning() << s; })
		.def("error", [](const std::string &s) { tlog::error() << s; })
		.def("success", [](const std::string &s) { tlog::success() << s; });
}

} // namespace ngp
