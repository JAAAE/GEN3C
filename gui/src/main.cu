/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/** @file   main.cu
 *  @author Thomas Müller, NVIDIA
 */

#include <neural-graphics-primitives/testbed.h>

#include <tiny-cuda-nn/common.h>

#include <args/args.hxx>

#include <filesystem/path.h>

using namespace args;
using namespace ngp;
using namespace std;

namespace ngp {

int main_func(const std::vector<std::string>& arguments) {
	ArgumentParser parser{
		"Gen3C GUI"
		"Version " NGP_VERSION,
		"",
	};

	HelpFlag help_flag{
		parser,
		"HELP",
		"Display this help menu.",
		{'h', "help"},
	};

	Flag vr_flag{parser, "VR", "Enables VR", {"vr"}};

	ValueFlag<string> snapshot_flag{
		parser,
		"SNAPSHOT",
		"Optional snapshot to load upon startup.",
		{"snapshot", "load_snapshot"},
	};

	ValueFlag<uint32_t> width_flag{
		parser,
		"WIDTH",
		"Resolution width of the GUI.",
		{"width"},
	};

	ValueFlag<uint32_t> height_flag{
		parser,
		"HEIGHT",
		"Resolution height of the GUI.",
		{"height"},
	};

	Flag version_flag{
		parser,
		"VERSION",
		"Display the version of Gen3C GUI.",
		{'v', "version"},
	};

	PositionalList<string> files{
		parser,
		"files",
		"Files to be loaded. Can be a scene, network config, snapshot, camera path, or a combination of those.",
	};

	// Parse command line arguments and react to parsing
	// errors using exceptions.
	try {
		if (arguments.empty()) {
			tlog::error() << "Number of arguments must be bigger than 0.";
			return -3;
		}

		parser.Prog(arguments.front());
		parser.ParseArgs(begin(arguments) + 1, end(arguments));
	} catch (const Help&) {
		cout << parser;
		return 0;
	} catch (const ParseError& e) {
		cerr << e.what() << endl;
		cerr << parser;
		return -1;
	} catch (const ValidationError& e) {
		cerr << e.what() << endl;
		cerr << parser;
		return -2;
	}

	if (version_flag) {
		tlog::none() << "Gen3C GUI v" NGP_VERSION;
		return 0;
	}

	Testbed testbed{ETestbedMode::Gen3c};

	for (auto file : get(files)) {
		testbed.load_file(file);
	}

#ifdef NGP_GUI
	bool gui = true;
#else
	bool gui = false;
#endif

	if (gui) {
		testbed.init_window(width_flag ? get(width_flag) : 1920, height_flag ? get(height_flag) : 1080);
	}

	if (vr_flag) {
		testbed.init_vr();
	}

	// Render loop
	while (testbed.frame()) {}

	return 0;
}

} // namespace ngp

#ifdef _WIN32
int wmain(int argc, wchar_t* argv[]) {
	SetConsoleOutputCP(CP_UTF8);
#else
int main(int argc, char* argv[]) {
#endif
	try {
		std::vector<std::string> arguments;
		for (int i = 0; i < argc; ++i) {
#ifdef _WIN32
			arguments.emplace_back(ngp::utf16_to_utf8(argv[i]));
#else
			arguments.emplace_back(argv[i]);
#endif
		}

		return ngp::main_func(arguments);
	} catch (const exception& e) {
		tlog::error() << fmt::format("Uncaught exception: {}", e.what());
		return 1;
	}
}
